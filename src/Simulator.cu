#include "hip/hip_runtime.h"
//
// Author: Francesco Arceri
// Date:   10-09-2021
//
// DEFINITION OF INTEGRATION FUNCTIONS

#include "../include/Simulator.h"
#include "../include/defs.h"
#include <thrust/transform.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <random>

using namespace std;
// position updates
__global__ void kernelUpdateParticlePos(double* pPos, const double* pVel, const double timeStep);
__global__ void kernelUpdateParticleVel(double* pVel, const double* pForce, const double timeStep);
// wall updates
__global__ void kernelUpdateWallPos(double* wPos, const double* wVel, const double timeStep);
__global__ void kernelUpdateWallVel(double* wVel, const double* wForce, const double timeStep);
// momentum conservation
__global__ void kernelSumParticleVelocity(double* pVel, double* velSum);
__global__ void kernelSubtractParticleDrift(double* pVel, double* velSum);
__global__ void kernelSubsetSumParticleVelocity(double* pVel, double* velSum, long firstId);
__global__ void kernelSubsetSubtractParticleDrift(double* pVel, double* velSum, long firstId);


//************************* soft particle langevin ***************************//
void SoftParticleLangevin::integrate() {
  updateVelocity(0.5 * sp_->dt);
  updatePosition(sp_->dt);
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  updateThermalVel();
  updateVelocity(0.5 * sp_->dt);
  conserveMomentum();
}

void SoftParticleLangevin::injectKineticEnergy() {
  double amplitude(sqrt(config.Tinject));
  // generate random numbers between 0 and noise for thermal noise
  thrust::counting_iterator<long> index_sequence_begin(lrand48());
  thrust::transform(index_sequence_begin, index_sequence_begin + sp_->numParticles * sp_->nDim, sp_->d_particleVel.begin(), gaussNum(0.f,amplitude));
  conserveMomentum();
}

void SoftParticleLangevin::updateThermalVel() {
  // generate random numbers between 0 and 1 for thermal noise
  thrust::counting_iterator<long> index_sequence_begin(lrand48());
  thrust::transform(index_sequence_begin, index_sequence_begin + sp_->numParticles * sp_->nDim, d_rand.begin(), gaussNum(0.f,1.f));
  // update thermal velocity
  long s_nDim(sp_->nDim);
  double s_gamma(gamma);
  double s_noise(noise);
  auto r = thrust::counting_iterator<long>(0);
  const double *rand = thrust::raw_pointer_cast(&d_rand[0]);
  const double *pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
  double *pForce = thrust::raw_pointer_cast(&(sp_->d_particleForce[0]));

  auto langevinAddThermostatForces = [=] __device__ (long particleId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pForce[particleId * s_nDim + dim] += s_noise * rand[particleId * s_nDim + dim] - s_gamma * pVel[particleId * s_nDim + dim];
    }
  };

  thrust::for_each(r, r + sp_->numParticles, langevinAddThermostatForces);
}

void SoftParticleLangevin::updateVelocity(double timeStep) {
  double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
	const double* pForce = thrust::raw_pointer_cast(&(sp_->d_particleForce[0]));
  kernelUpdateParticleVel<<<sp_->dimGrid, sp_->dimBlock>>>(pVel, pForce, timeStep);

  if(sp_->simControl.mobileType == simControlStruct::mobileEnum::on) {
    double* wVel = thrust::raw_pointer_cast(&(sp_->d_wallVel[0]));
    const double* wForce = thrust::raw_pointer_cast(&(sp_->d_wallForce[0]));
    kernelUpdateWallVel<<<sp_->dimGrid, sp_->dimBlock>>>(wVel, wForce, timeStep);
  }
}

void SoftParticleLangevin::updatePosition(double timeStep) {
	double* pPos = thrust::raw_pointer_cast(&(sp_->d_particlePos[0]));
	const double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
  kernelUpdateParticlePos<<<sp_->dimGrid, sp_->dimBlock>>>(pPos, pVel, timeStep);

  if(sp_->simControl.mobileType == simControlStruct::mobileEnum::on) {
    double* wPos = thrust::raw_pointer_cast(&(sp_->d_wallPos[0]));
    const double* wVel = thrust::raw_pointer_cast(&(sp_->d_wallVel[0]));
    kernelUpdateWallPos<<<sp_->dimGrid, sp_->dimBlock>>>(wPos, wVel, timeStep);
  }
}

void SoftParticleLangevin::conserveMomentum() {
  d_velSum.resize(sp_->nDim);
  thrust::fill(d_velSum.begin(), d_velSum.end(), double(0));
  double *velSum = thrust::raw_pointer_cast(&d_velSum[0]);
  double *pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
  kernelSumParticleVelocity<<<sp_->dimGrid, sp_->dimBlock>>>(pVel, velSum);
  kernelSubtractParticleDrift<<<sp_->dimGrid, sp_->dimBlock>>>(pVel, velSum);
}

//************************* soft particle langevin with driving force ***************************//
void SoftParticleDrivenLangevin::integrate() {
  updateVelocity(0.5 * sp_->dt);
  updatePosition(sp_->dt);
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  updateThermalVel();
  updateVelocity(0.5 * sp_->dt);
  //conserveMomentum();
}

void SoftParticleDrivenLangevin::updateThermalVel() {
  // update thermal velocity
  long s_nDim(sp_->nDim);
  double s_gamma(gamma);
  auto r = thrust::counting_iterator<long>(0);
  const double *pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
  double *pForce = thrust::raw_pointer_cast(&(sp_->d_particleForce[0]));

  auto langevinAddDampingForces = [=] __device__ (long particleId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pForce[particleId * s_nDim + dim] -= s_gamma * pVel[particleId * s_nDim + dim];
    }
  };

  thrust::for_each(r, r + sp_->numParticles, langevinAddDampingForces);
}

//************************* soft particle langevin ***************************//
void SoftParticleLangevin2::integrate() {
  updateThermalVel();
  updateVelocity(0.5*sp_->dt);
  updatePosition(sp_->dt);
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  updateVelocity(0.5*sp_->dt);
  conserveMomentum();
}

void SoftParticleLangevin2::updateThermalVel() {
  // extract two noises and compute noise terms
  thrust::counting_iterator<long> index_sequence_begin1(lrand48());
  thrust::transform(index_sequence_begin1, index_sequence_begin1 + sp_->numParticles * sp_->nDim, d_rand.begin(), gaussNum(0.f,1.f));
  thrust::counting_iterator<long> index_sequence_begin2(lrand48());
  thrust::transform(index_sequence_begin2, index_sequence_begin2 + sp_->numParticles * sp_->nDim, d_rando.begin(), gaussNum(0.f,1.f));
}

void SoftParticleLangevin2::updateVelocity(double timeStep) {
  long s_nDim(sp_->nDim);
  double s_dt(timeStep);
  double s_noise(noise);
  double s_gamma(gamma);
  auto r = thrust::counting_iterator<long>(0);
  const double *rand = thrust::raw_pointer_cast(&d_rand[0]);
  const double *rando = thrust::raw_pointer_cast(&d_rando[0]);
	double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
	const double* pForce = thrust::raw_pointer_cast(&(sp_->d_particleForce[0]));

  auto langevinUpdateParticleVel = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pVel[pId * s_nDim + dim] += s_dt * (pForce[pId * s_nDim + dim] - s_gamma * pVel[pId * s_nDim + dim] + s_noise * rand[pId * s_nDim + dim]) -
      s_dt * s_dt * s_gamma * 0.5 * (pForce[pId * s_nDim + dim] - s_gamma * pVel[pId * s_nDim + dim]) -
      s_dt * s_dt * s_gamma * s_noise * (0.5 * rand[pId * s_nDim + dim] + rando[pId * s_nDim + dim] / sqrt(3));
    }
  };

  thrust::for_each(r, r + sp_->numParticles, langevinUpdateParticleVel);
}

void SoftParticleLangevin2::updatePosition(double timeStep) {
  long s_nDim(sp_->nDim);
  double s_dt(timeStep);
  auto r = thrust::counting_iterator<long>(0);
  const double *rando = thrust::raw_pointer_cast(&d_rando[0]);
	double* pPos = thrust::raw_pointer_cast(&(sp_->d_particlePos[0]));
	const double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));

  auto langevinUpdateParticlePos = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pPos[pId * s_nDim + dim] += s_dt * pVel[pId * s_nDim + dim] + 0.5 * s_dt * s_dt * rando[pId * s_nDim + dim] / sqrt(3);
    }
  };

  thrust::for_each(r, r + sp_->numParticles, langevinUpdateParticlePos);
}

//************** soft particle langevin with massive particles ***************//
void SoftParticleLangevinSubSet::integrate() {
  updateThermalVel();
  updateVelocity(0.5*sp_->dt);
  updatePosition(sp_->dt);
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  updateVelocity(0.5*sp_->dt);
  //conserveMomentum();
}

void SoftParticleLangevinSubSet::updateThermalVel() {
  // extract two noises and compute noise terms
  thrust::counting_iterator<long> index_sequence_begin1(lrand48());
  thrust::transform(index_sequence_begin1, index_sequence_begin1 + sp_->numParticles * sp_->nDim, d_rand.begin(), gaussNum(0.f,1.f));
  thrust::counting_iterator<long> index_sequence_begin2(lrand48());
  thrust::transform(index_sequence_begin2, index_sequence_begin2 + sp_->numParticles * sp_->nDim, d_rando.begin(), gaussNum(0.f,1.f));
}

void SoftParticleLangevinSubSet::updateVelocity(double timeStep) {
  long s_nDim(sp_->nDim);
  double s_dt(timeStep);
  double s_gamma(gamma);
  double s_noise(noise);
  auto r = thrust::counting_iterator<long>(0);
  const double *rand = thrust::raw_pointer_cast(&d_rand[0]);
  const double *rando = thrust::raw_pointer_cast(&d_rando[0]);
	double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
	const double* pForce = thrust::raw_pointer_cast(&(sp_->d_particleForce[0]));

  auto langevinUpdateParticleVel = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pVel[pId * s_nDim + dim] += s_dt * (pForce[pId * s_nDim + dim] - s_gamma * pVel[pId * s_nDim + dim] + s_noise * rand[pId * s_nDim + dim]) -
      s_dt * s_dt * s_gamma * 0.5 * (pForce[pId * s_nDim + dim] - s_gamma * pVel[pId * s_nDim + dim]) -
      s_dt * s_dt * s_gamma * s_noise * (0.5 * rand[pId * s_nDim + dim] + rando[pId * s_nDim + dim] / sqrt(3));
    }
  };

  thrust::for_each(r + firstIndex, r + sp_->numParticles, langevinUpdateParticleVel);
  // update massive particles with normal langevin velocity update
  double s_mass(mass);
  auto s = thrust::counting_iterator<long>(0);
  auto updateMassiveParticleVel = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pVel[pId * s_nDim + dim] += s_dt * pForce[pId * s_nDim + dim] / s_mass;
    }
  };

  thrust::for_each(s, s + firstIndex, updateMassiveParticleVel);
}

void SoftParticleLangevinSubSet::updatePosition(double timeStep) {
  long s_nDim(sp_->nDim);
  double s_dt(timeStep);
  auto r = thrust::counting_iterator<long>(0);
  const double *rando = thrust::raw_pointer_cast(&d_rando[0]);
	double* pPos = thrust::raw_pointer_cast(&(sp_->d_particlePos[0]));
	const double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));

  auto langevinUpdateParticlePos = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pPos[pId * s_nDim + dim] += s_dt * pVel[pId * s_nDim + dim] + 0.5 * s_dt * s_dt * rando[pId * s_nDim + dim] / sqrt(3);
    }
  };

  thrust::for_each(r + firstIndex, r + sp_->numParticles, langevinUpdateParticlePos);
  // update massive particles with normal langevin position update
  auto s = thrust::counting_iterator<long>(0);
  auto updateMassiveParticlePos = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pPos[pId * s_nDim + dim] += s_dt * pVel[pId * s_nDim + dim];
    }
  };

  thrust::for_each(s, s + firstIndex, updateMassiveParticlePos);
}

void SoftParticleLangevinSubSet::conserveMomentum() {
  d_velSum.resize(sp_->nDim);
  thrust::fill(d_velSum.begin(), d_velSum.end(), double(0));
  double *velSum = thrust::raw_pointer_cast(&d_velSum[0]);
  double *pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
  kernelSubsetSumParticleVelocity<<<sp_->dimGrid, sp_->dimBlock>>>(pVel, velSum, firstIndex);
  kernelSubsetSubtractParticleDrift<<<sp_->dimGrid, sp_->dimBlock>>>(pVel, velSum, firstIndex);
}

//*************** soft particle langevin with external field *****************//
void SoftParticleLangevinExtField::integrate() {
  updateThermalVel();
  updateVelocity(0.5*sp_->dt);
  updatePosition(sp_->dt);
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  sp_->addExternalParticleForce();
  updateVelocity(0.5*sp_->dt);
  //conserveMomentum();
}

//******* soft particle langevin with perturbation on first particles ********//
void SoftParticleLangevinPerturb::integrate() {
  updateThermalVel();
  updateVelocity(0.5*sp_->dt);
  updatePosition(sp_->dt);
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  sp_->addConstantParticleForce(extForce, firstIndex);
  updateVelocity(0.5*sp_->dt);
  //conserveMomentum();
}

//****************** soft particle langevin with fluid flow ******************//
void SoftParticleLangevinFlow::integrate() {
  updateThermalVel();
  updateVelocity(0.5*sp_->dt);
  updatePosition(sp_->dt);
  sp_->calcFlowVelocity();
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  updateVelocity(0.5*sp_->dt);
  //conserveMomentum();
}

void SoftParticleLangevinFlow::updateVelocity(double timeStep) {
  long s_nDim(sp_->nDim);
  double s_dt(timeStep);
  double s_gamma(gamma);
  double s_noise(noise);
  auto r = thrust::counting_iterator<long>(0);
  const double *rand = thrust::raw_pointer_cast(&d_rand[0]);
  const double *rando = thrust::raw_pointer_cast(&d_rando[0]);
  const double *flowVel = thrust::raw_pointer_cast(&(sp_->d_flowVel[0]));
	double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
	const double* pForce = thrust::raw_pointer_cast(&(sp_->d_particleForce[0]));

  auto langevinUpdateParticleFlowVel = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pVel[pId * s_nDim + dim] += s_dt * (pForce[pId * s_nDim + dim] + s_gamma * (flowVel[pId * s_nDim + dim] - pVel[pId * s_nDim + dim]) + s_noise * rand[pId * s_nDim + dim]) -
      s_dt * s_dt * s_gamma * 0.5 * (pForce[pId * s_nDim + dim] + s_gamma * (flowVel[pId * s_nDim + dim] - pVel[pId * s_nDim + dim])) -
      s_dt * s_dt * s_gamma * s_noise * (0.5 * rand[pId * s_nDim + dim] + rando[pId * s_nDim + dim] / sqrt(3));
    }
  };


  thrust::for_each(r, r + sp_->numParticles, langevinUpdateParticleFlowVel);
  //kernelConserveParticleMomentum<<<1, sp_->dimBlock>>>(pVel);
}

//*************** soft particle damped dynamics with fluid flow ***************//
void SoftParticleFlow::integrate() {
  updateVelocity(0.5*sp_->dt);
  updatePosition(sp_->dt);
  sp_->calcFlowVelocity();
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  updateVelocity(0.5*sp_->dt);
  //conserveMomentum();
}

void SoftParticleFlow::updateVelocity(double timeStep) {
  long s_nDim(sp_->nDim);
  double s_dt(timeStep);
  double s_gamma(gamma);
  auto r = thrust::counting_iterator<long>(0);
  const double *flowVel = thrust::raw_pointer_cast(&(sp_->d_flowVel[0]));
	double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
	const double* pForce = thrust::raw_pointer_cast(&(sp_->d_particleForce[0]));

  auto updateParticleFlowVel = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pVel[pId * s_nDim + dim] += s_dt * (pForce[pId * s_nDim + dim] + s_gamma * (flowVel[pId * s_nDim + dim] - pVel[pId * s_nDim + dim]));
      pVel[pId * s_nDim + dim] -= 0.5 * s_dt * s_dt * s_gamma * (pForce[pId * s_nDim + dim] + s_gamma * (flowVel[pId * s_nDim + dim] - pVel[pId * s_nDim + dim]));
    }
  };

  thrust::for_each(r, r + sp_->numParticles, updateParticleFlowVel);
  //kernelConserveParticleMomentum<<<1, sp_->dimBlock>>>(pVel);
}

void SoftParticleFlow::updatePosition(double timeStep) {
  long s_nDim(sp_->nDim);
  double s_dt(timeStep);
  auto r = thrust::counting_iterator<long>(0);
	double* pPos = thrust::raw_pointer_cast(&(sp_->d_particlePos[0]));
	const double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));

  auto updateParticleFlowPos = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pPos[pId * s_nDim + dim] += s_dt * pVel[pId * s_nDim + dim];
    }
  };

  thrust::for_each(r, r + sp_->numParticles, updateParticleFlowPos);
}

//**************************** soft particle nve *****************************//
void SoftParticleNVE::integrate() {
  updateVelocity(0.5 * sp_->dt);
  updatePosition(sp_->dt);
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  updateVelocity(0.5 * sp_->dt);
  sp_->checkReflectiveWall();
  //conserveMomentum();
}

//**************** soft particle nve with velocity rescaling *****************//
void SoftParticleNVERescale::integrate() {
  injectKineticEnergy();
  updateVelocity(0.5 * sp_->dt);
  updatePosition(sp_->dt);
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  updateVelocity(0.5 * sp_->dt);
  //conserveMomentum();
}

void SoftParticleNVERescale::injectKineticEnergy() {
  double scale = sqrt(config.Tinject / sp_->getParticleTemperature());
  long s_nDim(sp_->nDim);
  auto r = thrust::counting_iterator<long>(0);
	double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));

  auto scaleParticleVel = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pVel[pId * s_nDim + dim] *= scale;
    }
  };

  thrust::for_each(r, r + sp_->numParticles, scaleParticleVel);
}

//**************** soft particle nve with velocity rescaling *****************//
void SoftParticleNVEDoubleRescale::integrate() {
  injectKineticEnergy();
  updateVelocity(0.5 * sp_->dt);
  updatePosition(sp_->dt);
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  updateVelocity(0.5 * sp_->dt);
  //conserveMomentum();
}

void SoftParticleNVEDoubleRescale::injectKineticEnergy() {
  std::tuple<double, double, double> Temps = sp_->getParticleT1T2();
  double scale1 = sqrt(config.Tinject / get<0>(Temps));
  double scale2 = sqrt(config.driving / get<1>(Temps));
  long s_nDim(sp_->nDim);
  long s_num1(sp_->num1);
  auto r = thrust::counting_iterator<long>(0);
	double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));

  auto doubleScaleParticleVel = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      if(pId < s_num1) {
        pVel[pId * s_nDim + dim] *= scale1;
      } else {
        pVel[pId * s_nDim + dim] *= scale2;
      }
    }
  };

  thrust::for_each(r, r + sp_->num1, doubleScaleParticleVel);
}

//************************ soft particle Nose Hoover **************************//
void SoftParticleNoseHoover::integrate() {
  updateVelocity(0.5 * sp_->dt);
  updatePosition(sp_->dt);
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  updateThermalVel();
}

void SoftParticleNoseHoover::updateVelocity(double timeStep) {
  // update nose hoover damping
  gamma += (sp_->dt / (2 * mass)) * (sp_->getParticleKineticEnergy() - (sp_->nDim * sp_->numParticles + 1) * config.Tinject / 2);
  double s_gamma(gamma);
  long s_nDim(sp_->nDim);
  double s_dt(timeStep);
  auto r = thrust::counting_iterator<long>(0);
	double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
	const double* pForce = thrust::raw_pointer_cast(&(sp_->d_particleForce[0]));

  auto noseHooverUpdateParticleVel = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pVel[pId * s_nDim + dim] += s_dt * (pForce[pId * s_nDim + dim] - pVel[pId * s_nDim + dim] * s_gamma);
    }
  };

  thrust::for_each(r, r + sp_->numParticles, noseHooverUpdateParticleVel);
}

void SoftParticleNoseHoover::updateThermalVel() {
  // update nose hoover damping
  gamma += (sp_->dt / (2 * mass)) * (sp_->getParticleKineticEnergy() - (sp_->nDim * sp_->numParticles + 1) * config.Tinject / 2);
  double s_gamma(gamma);
  long s_nDim(sp_->nDim);
  double s_dt(sp_->dt);
  auto r = thrust::counting_iterator<long>(0);
	double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
	const double* pForce = thrust::raw_pointer_cast(&(sp_->d_particleForce[0]));

  auto noseHooverSecondUpdateParticleVel = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      pVel[pId * s_nDim + dim] = (pVel[pId * s_nDim + dim] + 0.5 * s_dt * pForce[pId * s_nDim + dim]) / (1 + 0.5 * s_dt * s_gamma);
    }
  };

  thrust::for_each(r, r + sp_->numParticles, noseHooverSecondUpdateParticleVel);
}

//******************** soft particle double T Nose Hoover *******************//
void SoftParticleDoubleNoseHoover::integrate() {
  updateVelocity(0.5 * sp_->dt);
  updatePosition(sp_->dt);
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  updateThermalVel();
}

void SoftParticleDoubleNoseHoover::injectKineticEnergy() {
  double amplitude1(sqrt(config.Tinject));
  double amplitude2(sqrt(config.driving));
  // generate random numbers between 0 and noise for thermal noise
  thrust::counting_iterator<long> index_sequence_begin1(lrand48());
  thrust::transform(index_sequence_begin1, index_sequence_begin1 + sp_->num1 * sp_->nDim, sp_->d_particleVel.begin(), gaussNum(0.f,amplitude1));
  thrust::counting_iterator<long> index_sequence_begin2(lrand48());
  thrust::transform(index_sequence_begin2, index_sequence_begin2 + (sp_->numParticles - sp_->num1) * sp_->nDim, sp_->d_particleVel.begin() + sp_->num1 * sp_->nDim, gaussNum(0.f,amplitude2));
  conserveMomentum();
}

void SoftParticleDoubleNoseHoover::updateVelocity(double timeStep) {
  // update nose hoover damping
  std::tuple<double, double, double> ekins = sp_->getParticleKineticEnergy12();
  gamma1 += (sp_->dt / (2 * mass)) * (get<0>(ekins) - (sp_->nDim * sp_->num1 + 1) * config.Tinject / 2);//T1
  gamma2 += (sp_->dt / (2 * mass)) * (get<1>(ekins) - (sp_->nDim * (sp_->numParticles - sp_->num1) + 1) * config.driving / 2);//T2
  double s_gamma1(gamma1);
  double s_gamma2(gamma2);
  long s_nDim(sp_->nDim);
  long s_num1(sp_->num1);
  double s_dt(timeStep);
  auto r = thrust::counting_iterator<long>(0);
	double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
	const double* pForce = thrust::raw_pointer_cast(&(sp_->d_particleForce[0]));

  auto doubleNoseHooverUpdateParticleVel = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      if(pId < s_num1) {
        pVel[pId * s_nDim + dim] += s_dt * (pForce[pId * s_nDim + dim] - pVel[pId * s_nDim + dim] * s_gamma1);
      } else {
        pVel[pId * s_nDim + dim] += s_dt * (pForce[pId * s_nDim + dim] - pVel[pId * s_nDim + dim] * s_gamma2);
      }
    }
  };

  thrust::for_each(r, r + sp_->numParticles, doubleNoseHooverUpdateParticleVel);
}

void SoftParticleDoubleNoseHoover::updateThermalVel() {
  // update nose hoover damping
  std::tuple<double, double, double> ekins = sp_->getParticleKineticEnergy12();
  gamma1 += (sp_->dt / (2 * mass)) * (get<0>(ekins) - (sp_->nDim * sp_->num1 + 1) * config.Tinject / 2);//T1
  gamma2 += (sp_->dt / (2 * mass)) * (get<1>(ekins) - (sp_->nDim * (sp_->numParticles - sp_->num1) + 1) * config.driving / 2);//T2
  double s_gamma1(gamma1);
  double s_gamma2(gamma2);
  long s_nDim(sp_->nDim);
  long s_num1(sp_->num1);
  double s_dt(sp_->dt);
  auto r = thrust::counting_iterator<long>(0);
	double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
	const double* pForce = thrust::raw_pointer_cast(&(sp_->d_particleForce[0]));

  auto doubleNoseHooverSecondUpdateParticleVel = [=] __device__ (long pId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      if(pId < s_num1) {
        pVel[pId * s_nDim + dim] = (pVel[pId * s_nDim + dim] + 0.5 * s_dt * pForce[pId * s_nDim + dim]) / (1 + 0.5 * s_dt * s_gamma1);
      } else {
        pVel[pId * s_nDim + dim] = (pVel[pId * s_nDim + dim] + 0.5 * s_dt * pForce[pId * s_nDim + dim]) / (1 + 0.5 * s_dt * s_gamma2);
      }
    }
  };

  thrust::for_each(r, r + sp_->numParticles, doubleNoseHooverSecondUpdateParticleVel);
}

//**************************** brownian integrator *****************************//
void SoftParticleBrownian::integrate() {
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  sp_->checkReflectiveWall();
  updatePosition(sp_->dt);
  //conserveMomentum();
}

void SoftParticleBrownian::updateThermalVel() {
  // generate random numbers between 0 and 1 for thermal noise
  thrust::counting_iterator<long> index_sequence_begin(lrand48());
  thrust::transform(index_sequence_begin, index_sequence_begin + sp_->numParticles * sp_->nDim, d_rand.begin(), gaussNum(0.f,1.f));
  // assign overdamped velocity
  long s_nDim(sp_->nDim);
  double s_gamma(gamma);
  double s_noise(noise);
  auto r = thrust::counting_iterator<long>(0);
  const double *rand = thrust::raw_pointer_cast(&d_rand[0]);
  const double *pForce = thrust::raw_pointer_cast(&(sp_->d_particleForce[0]));
  double *pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));

  auto updateBrownianVel = [=] __device__ (long particleId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
		  pVel[particleId * s_nDim + dim] = (pForce[particleId * s_nDim + dim] + s_noise * rand[particleId * s_nDim + dim]) / s_gamma;
    }
  };

  thrust::for_each(r, r + sp_->numParticles, updateBrownianVel);
}

void SoftParticleBrownian::updatePosition(double timeStep) {
	double* pPos = thrust::raw_pointer_cast(&(sp_->d_particlePos[0]));
	const double* pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));
  kernelUpdateParticlePos<<<sp_->dimGrid, sp_->dimBlock>>>(pPos, pVel, timeStep);
}

//**************************** driven brownian integrator *****************************//
void SoftParticleDrivenBrownian::integrate() {
  sp_->checkParticleNeighbors();
  sp_->calcParticleForceEnergy();
  updateThermalVel();
  sp_->checkReflectiveWall();
  updatePosition(sp_->dt);
  //conserveMomentum();
}

void SoftParticleDrivenBrownian::updateThermalVel() {
  // assign overdamped velocity as total force over damping
  long s_nDim(sp_->nDim);
  double s_gamma(gamma);
  auto r = thrust::counting_iterator<long>(0);
  const double *pForce = thrust::raw_pointer_cast(&(sp_->d_particleForce[0]));
  double *pVel = thrust::raw_pointer_cast(&(sp_->d_particleVel[0]));

  auto updateActiveBrownianVel = [=] __device__ (long particleId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < s_nDim; dim++) {
      // self-propulsion has already been added to the force
		  pVel[particleId * s_nDim + dim] = pForce[particleId * s_nDim + dim] / s_gamma;
    }
  };

  thrust::for_each(r, r + sp_->numParticles, updateActiveBrownianVel);
}
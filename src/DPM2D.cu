#include "hip/hip_runtime.h"
//
// Author: Francesco Arceri
// Date:   10-03-2021
//
// FUNCTION DECLARATIONS

#include "../include/DPM2D.h"
#include "../include/DPM2DKernel.cuh"
#include "../include/Simulator.h"
#include "../include/FIRE.h"
#include <iostream>
#include <string>
#include <vector>
#include <cmath>
#include <random>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>

using namespace std;
using std::cout;
using std::endl;

//************************** dpm object definition ***************************//
DPM2D::DPM2D(long nParticles, long dim, long nVertexPerParticle) {
  // default values
  srand48(time(0));
  dimBlock = 256;
  nDim = dim;
  numParticles = nParticles;
  numVertexPerParticle = nVertexPerParticle;
  // the default is monodisperse size distribution
  // same number of vertices per particle
  numVertices = numParticles * numVertexPerParticle;
  setDimBlock(dimBlock);
  setNDim(nDim);
  setNumParticles(numParticles);
  setNumVertexPerParticle(numVertexPerParticle);
  setNumVertices(numVertices);
  // set force paramters to zero
  // TODO: maybe initialize these?
  dt = 0.;
  rho0 = 0.;
  ea = 0.;
	el = 0.;
	eb = 0.;
	ec = 0.;
	l1 = 0.;
	l2 = 0.;
  d_boxSize.resize(nDim);
  thrust::fill(d_boxSize.begin(), d_boxSize.end(), double(1));
  d_stress.resize(nDim * nDim);
  thrust::fill(d_stress.begin(), d_stress.end(), double(0));
  d_numVertexInParticleList.resize(numParticles);
  setMonoSizeDistribution();
  d_firstVertexInParticleId.resize(numParticles);
  initParticleIdList();
  // particle variables
  initParticleVariables(numParticles);
  // particle dynmaical variables
  initParticleDynamicalVariables(numParticles);
  // vertex shape variables
  initVertexVariables(numVertices);
  // vertex dynamical variables
  initDynamicalVariables(numVertices);
  // initialize contacts and neighbors
  initContacts(numParticles);
  initNeighbors(numVertices);
  syncNeighborsToDevice();
  initParticleNeighbors(numParticles);
  syncParticleNeighborsToDevice();
}

DPM2D::~DPM2D() {
	// clear all vectors and pointers
	d_boxSize.clear();
  d_stress.clear();
  d_numVertexInParticleList.clear();
  d_firstVertexInParticleId.clear();
  d_particleIdList.clear();
  d_a0.clear();
  d_rad.clear();
  d_l0.clear();
  d_theta0.clear();
  d_length.clear();
  d_l0Vel.clear();
  d_area.clear();
  d_perimeter.clear();
  d_particleRad.clear();
  d_particlePos.clear();
  d_particleVel.clear();
  d_particleForce.clear();
  d_particleEnergy.clear();
  d_particleTorque.clear();
  d_particleAngvel.clear();
  d_particleAngle.clear();
  d_particleInitAngle.clear();
  // dynamical variables
  d_pos.clear();
  d_vel.clear();
  d_force.clear();
  d_energy.clear();
  d_torque.clear();
  d_initialPos.clear();
  d_particleInitPos.clear();
  d_particleDelta.clear();
  d_particleDeltaAngle.clear();
  d_particlePreviousPos.clear();
  // contacts and neighbors
  d_contactList.clear();
  d_numContacts.clear();
  d_contactVectorList.clear();
  d_neighborList.clear();
  d_maxNeighborList.clear();
  d_numPartNeighbors.clear();
  d_partNeighborList.clear();
  d_partMaxNeighborList.clear();
}

void DPM2D::initParticleVariables(long numParticles_) {
  d_a0.resize(numParticles_);
  d_area.resize(numParticles_);
  d_perimeter.resize(numParticles_);
  d_particleAngle.resize(numParticles_);
  d_perParticleStress.resize(numParticles_ * nDim * nDim);
  thrust::fill(d_a0.begin(), d_a0.end(), double(0));
  thrust::fill(d_area.begin(), d_area.end(), double(0));
  thrust::fill(d_perimeter.begin(), d_perimeter.end(), double(0));
  thrust::fill(d_particleAngle.begin(), d_particleAngle.end(), double(0));
  thrust::fill(d_perParticleStress.begin(), d_perParticleStress.end(), double(0));
}

void DPM2D::initParticleDynamicalVariables(long numParticles_) {
  d_particleRad.resize(numParticles_);
  d_particlePos.resize(numParticles_ * nDim);
  d_particleVel.resize(numParticles_ * nDim);
  d_particleForce.resize(numParticles_ * nDim);
  d_particleEnergy.resize(numParticles_);
  thrust::fill(d_particleRad.begin(), d_particleRad.end(), double(0));
  thrust::fill(d_particlePos.begin(), d_particlePos.end(), double(0));
  thrust::fill(d_particleVel.begin(), d_particleVel.end(), double(0));
  thrust::fill(d_particleForce.begin(), d_particleForce.end(), double(0));
  thrust::fill(d_particleEnergy.begin(), d_particleEnergy.end(), double(0));
}

void DPM2D::initRotationalVariables(long numVertices_, long numParticles_) {
  d_torque.resize(numVertices_);
  d_particleAngvel.resize(numParticles_);
  d_particleTorque.resize(numParticles_);
  thrust::fill(d_torque.begin(), d_torque.end(), double(0));
  thrust::fill(d_particleAngvel.begin(), d_particleAngvel.end(), double(0));
  thrust::fill(d_particleTorque.begin(), d_particleTorque.end(), double(0));
}

void DPM2D::initVertexVariables(long numVertices_) {
  d_rad.resize(numVertices_);
  d_l0.resize(numVertices_);
  d_theta0.resize(numVertices_);
  d_length.resize(numVertices_);
  thrust::fill(d_rad.begin(), d_rad.end(), double(0));
  thrust::fill(d_l0.begin(), d_l0.end(), double(0));
  thrust::fill(d_theta0.begin(), d_theta0.end(), double(0));
  thrust::fill(d_length.begin(), d_length.end(), double(0));
}

void DPM2D::initDynamicalVariables(long numVertices_) {
  d_pos.resize(numVertices_ * nDim);
  d_vel.resize(numVertices_ * nDim);
  d_force.resize(numVertices_ * nDim);
  d_energy.resize(numVertices_);
  d_lastPos.resize(numVertices_ * nDim);
  d_disp.resize(numVertices_);
  thrust::fill(d_pos.begin(), d_pos.end(), double(0));
  thrust::fill(d_vel.begin(), d_vel.end(), double(0));
  thrust::fill(d_force.begin(), d_force.end(), double(0));
  thrust::fill(d_energy.begin(), d_energy.end(), double(0));
  thrust::fill(d_lastPos.begin(), d_lastPos.end(), double(0));
  thrust::fill(d_disp.begin(), d_disp.end(), double(0));
}

void DPM2D::initDeltaVariables(long numVertices_, long numParticles_) {
  d_initialPos.resize(numVertices_ * nDim);
  d_delta.resize(numVertices_ * nDim);
  d_particleInitPos.resize(numParticles_ * nDim);
  d_particleDelta.resize(numParticles_ * nDim);
  d_particleDisp.resize(numParticles_);
  d_particleInitAngle.resize(numParticles_);
  d_particleDeltaAngle.resize(numParticles_);
  thrust::fill(d_initialPos.begin(), d_initialPos.end(), double(0));
  thrust::fill(d_delta.begin(), d_delta.end(), double(0));
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  thrust::fill(d_particleDelta.begin(), d_particleDelta.end(), double(0));
  thrust::fill(d_particleDisp.begin(), d_particleDisp.end(), double(0));
  thrust::fill(d_particleInitAngle.begin(), d_particleInitAngle.end(), double(0));
  thrust::fill(d_particleDeltaAngle.begin(), d_particleDeltaAngle.end(), double(0));
}

void DPM2D::initContacts(long numParticles_) {
  long maxContacts = 8 * nDim; // guess
  d_numContacts.resize(numParticles_);
  d_contactList.resize(numParticles_ * maxContacts);
  d_numPartNeighbors.resize(numParticles_);
  d_partNeighborList.resize(numParticles_ * maxContacts);
  d_contactVectorList.resize(numParticles_ * nDim * maxContacts);
  thrust::fill(d_numContacts.begin(), d_numContacts.end(), -1L);
  thrust::fill(d_contactList.begin(), d_contactList.end(), double(0));
  thrust::fill(d_numPartNeighbors.begin(), d_numPartNeighbors.end(), -1L);
  thrust::fill(d_partNeighborList.begin(), d_partNeighborList.end(), double(0));
  thrust::fill(d_contactVectorList.begin(), d_contactVectorList.end(), double(0));
}

void DPM2D::initNeighbors(long numVertices_) {
  neighborListSize = 0;
  maxNeighbors = 0;
  d_neighborList.resize(numVertices_);
  d_maxNeighborList.resize(numVertices_);
  thrust::fill(d_neighborList.begin(), d_neighborList.end(), -1L);
  thrust::fill(d_maxNeighborList.begin(), d_maxNeighborList.end(), maxNeighbors);
}

void DPM2D::initParticleNeighbors(long numParticles_) {
  partNeighborListSize = 0;
  partMaxNeighbors = 0;
  d_partNeighborList.resize(numParticles_);
  d_partMaxNeighborList.resize(numParticles_);
  thrust::fill(d_partNeighborList.begin(), d_partNeighborList.end(), -1L);
  thrust::fill(d_partMaxNeighborList.begin(), d_partMaxNeighborList.end(), partMaxNeighbors);
}


void DPM2D::initParticleIdList() {
  long countVertices = 0;
  d_particleIdList.resize(numVertices);
  for (long particleId = 0; particleId < numParticles; particleId++) {
    d_firstVertexInParticleId[particleId] = countVertices;
    for(long vertexInPartId = 0; vertexInPartId < d_numVertexInParticleList[particleId]; vertexInPartId++) {
      d_particleIdList[countVertices] = particleId;
			countVertices += 1;
		}
  }
  long* firstVertexInParticleId = thrust::raw_pointer_cast(&d_firstVertexInParticleId[0]);
  hipMemcpyToSymbol(HIP_SYMBOL(d_firstVertexInParticleIdPtr), &firstVertexInParticleId, sizeof(firstVertexInParticleId));

  long* particleIdList = thrust::raw_pointer_cast(&d_particleIdList[0]);
  hipMemcpyToSymbol(HIP_SYMBOL(d_particleIdListPtr), &particleIdList, sizeof(particleIdList));
}

//**************************** setters and getters ***************************//
// TODO: add error checks for all the getters and setters
void DPM2D::setDimBlock(long dimBlock_) {
	dimBlock = dimBlock_;
	dimGrid = (numVertices + dimBlock - 1) / dimBlock;
	partDimGrid = (numParticles + dimBlock - 1) / dimBlock;
  hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(d_dimBlock), &dimBlock, sizeof(dimBlock));
  if(err != hipSuccess) {
    cout << "hipMemcpyToSymbol Error: "<< hipGetErrorString(err) << endl;
  }
  err = hipMemcpyToSymbol(HIP_SYMBOL(d_dimGrid), &dimGrid, sizeof(dimGrid));
  if(err != hipSuccess) {
    cout << "hipMemcpyToSymbol Error: "<< hipGetErrorString(err) << endl;
  }
  err = hipMemcpyToSymbol(HIP_SYMBOL(d_partDimGrid), &partDimGrid, sizeof(partDimGrid));
  if(err != hipSuccess) {
    cout << "hipMemcpyToSymbol Error: "<< hipGetErrorString(err) << endl;
  }
}

long DPM2D::getDimBlock() {
  long dimBlockFromDevice;
  //dimBlockFromDevice = d_dimBlock;
  hipError_t err = hipMemcpyFromSymbol(&dimBlockFromDevice, HIP_SYMBOL(d_dimBlock), sizeof(d_dimBlock));
  if(err != hipSuccess) {
    cout << "hipMemcpyToSymbol Error: "<< hipGetErrorString(err) << endl;
  }
  if (dimBlock != dimBlockFromDevice) {
    cout << "DPM::getDimBlock: dimBlock on host does not match dimBlock on device" << endl;
  }
	return dimBlockFromDevice;
}

void DPM2D::setNDim(long nDim_) {
  nDim = nDim_;
  hipMemcpyToSymbol(HIP_SYMBOL(d_nDim), &nDim, sizeof(nDim));
}

long DPM2D::getNDim() {
  long nDimFromDevice;
  hipMemcpyFromSymbol(&nDimFromDevice, HIP_SYMBOL(d_nDim), sizeof(d_nDim));
	return nDimFromDevice;
}

void DPM2D::setNumParticles(long numParticles_) {
  numParticles = numParticles_;
  hipMemcpyToSymbol(HIP_SYMBOL(d_numParticles), &numParticles, sizeof(numParticles));
}

long DPM2D::getNumParticles() {
  long numParticlesFromDevice;
  hipMemcpyFromSymbol(&numParticlesFromDevice, HIP_SYMBOL(d_numParticles), sizeof(d_numParticles));
	return numParticlesFromDevice;
}

void DPM2D::setNumVertices(long numVertices_) {
  numVertices = numVertices_;
  hipMemcpyToSymbol(HIP_SYMBOL(d_numVertices), &(numVertices), sizeof(numVertices));
  setDimBlock(dimBlock); // recalculate dimGrid
}

long DPM2D::getNumVertices() {
  long numVerticesFromDevice;
  hipMemcpyFromSymbol(&numVerticesFromDevice, HIP_SYMBOL(d_numVertices), sizeof(d_numVertices));
	return numVerticesFromDevice;
}

void DPM2D::setNumVertexPerParticle(long numVertexPerParticle_) {
  numVertexPerParticle = numVertexPerParticle_;
  hipMemcpyToSymbol(HIP_SYMBOL(d_numVertexPerParticle), &numVertexPerParticle, sizeof(numVertexPerParticle));
}

long DPM2D::getNumVertexPerParticle() {
  long numVertexPerParticleFromDevice;
  hipMemcpyFromSymbol(&numVertexPerParticleFromDevice, HIP_SYMBOL(d_numVertexPerParticle), sizeof(d_numVertexPerParticle));
  return numVertexPerParticleFromDevice;
}

void DPM2D::setNumVertexInParticleList(thrust::host_vector<long> &numVertexInParticleList_) {
  if(numVertexInParticleList_.size() == ulong(numParticles)) {
    d_numVertexInParticleList = numVertexInParticleList_;
    long* numVertexInParticleList = thrust::raw_pointer_cast(&(d_numVertexInParticleList[0]));
    hipMemcpyToSymbol(HIP_SYMBOL(d_numVertexInParticleListPtr), &numVertexInParticleList, sizeof(numVertexInParticleList));
  } else {
    cout << "DPM2D::setNumVertexInParticleList: size of numVertexInParticleList does not match numParticles" << endl;
  }
}

thrust::host_vector<long> DPM2D::getNumVertexInParticleList() {
  thrust::host_vector<long> numVertexInParticleListFromDevice;
  if(d_numVertexInParticleList.size() == ulong(numParticles)) {
    hipMemcpyFromSymbol(&d_numVertexInParticleList, HIP_SYMBOL(d_numVertexInParticleListPtr), sizeof(d_numVertexInParticleListPtr));
    numVertexInParticleListFromDevice = d_numVertexInParticleList;
  } else {
    cout << "DPM2D::getNumVertexInParticleList: size of numVertexInParticleList from device does not match numParticles" << endl;
  }
  return numVertexInParticleListFromDevice;
}

// the length scale is always set to be the sqrt of the first particle area
void DPM2D::setLengthScale() {
  rho0 = sqrt((thrust::reduce(d_a0.begin(), d_a0.end(), double(0), thrust::plus<double>()))/numParticles); // set dimensional factor
  //cout << " lengthscale: " << rho0 << endl;
  hipMemcpyToSymbol(HIP_SYMBOL(d_rho0), &rho0, sizeof(rho0));
}

void DPM2D::setParticleLengthScale() {
  rho0 = thrust::reduce(d_particleRad.begin(), d_particleRad.end(), double(0), thrust::plus<double>())/numParticles; // set dimensional factor
  cout << " lengthscale: " << rho0 << endl;
  hipMemcpyToSymbol(HIP_SYMBOL(d_rho0), &rho0, sizeof(rho0));
}

void DPM2D::setLengthScaleToOne() {
  rho0 = 1.; // for soft particles
  hipMemcpyToSymbol(HIP_SYMBOL(d_rho0), &rho0, sizeof(rho0));
}

//TODO: error messages for all the vector getters and setters
void DPM2D::setBoxSize(thrust::host_vector<double> &boxSize_) {
  if(boxSize_.size() == ulong(nDim)) {
    d_boxSize = boxSize_;
    double* boxSize = thrust::raw_pointer_cast(&(d_boxSize[0]));
    hipMemcpyToSymbol(HIP_SYMBOL(d_boxSizePtr), &boxSize, sizeof(boxSize));
  } else {
    cout << "DPM2D::setBoxSize: size of boxSize does not match nDim" << endl;
  }
}

thrust::host_vector<double> DPM2D::getBoxSize() {
  thrust::host_vector<double> boxSizeFromDevice;
  if(d_boxSize.size() == ulong(nDim)) {
    hipMemcpyFromSymbol(&d_boxSize, HIP_SYMBOL(d_boxSizePtr), sizeof(d_boxSizePtr));
    boxSizeFromDevice = d_boxSize;
  } else {
    cout << "DPM2D::getBoxSize: size of boxSize from device does not match nDim" << endl;
  }
  return boxSizeFromDevice;
}

//**************************** shape variables *******************************//
void DPM2D::setVertexRadii(thrust::host_vector<double> &rad_) {
  d_rad = rad_;
}

thrust::host_vector<double> DPM2D::getVertexRadii() {
  thrust::host_vector<double> radFromDevice;
  radFromDevice = d_rad;
  return radFromDevice;
}

double DPM2D::getMaxRadius() {
  return double(thrust::reduce(d_rad.begin(), d_rad.end(), double(-1), thrust::maximum<double>()));
}

void DPM2D::setRestAreas(thrust::host_vector<double> &a0_) {
  d_a0 = a0_;
}

thrust::host_vector<double> DPM2D::getRestAreas() {
  thrust::host_vector<double> a0FromDevice;
  a0FromDevice = d_a0;
  return a0FromDevice;
}

void DPM2D::setRestLengths(thrust::host_vector<double> &l0_) {
  d_l0 = l0_;
}

thrust::host_vector<double> DPM2D::getRestLengths() {
  thrust::host_vector<double> l0FromDevice;
  l0FromDevice = d_l0;
  return l0FromDevice;
}

void DPM2D::setRestAngles(thrust::host_vector<double> &theta0_) {
  d_theta0 = theta0_;
}

thrust::host_vector<double> DPM2D::getRestAngles() {
  thrust::host_vector<double> theta0FromDevice;
  theta0FromDevice = d_theta0;
  return theta0FromDevice;
}

thrust::host_vector<double> DPM2D::getSegmentLengths() {
  thrust::host_vector<double> lengthFromDevice;
  lengthFromDevice = d_length;
  return lengthFromDevice;
}

void DPM2D::setAreas(thrust::host_vector<double> &area_) {
  d_area = area_;
}

thrust::host_vector<double> DPM2D::getAreas() {
  thrust::host_vector<double> areaFromDevice;
  areaFromDevice = d_area;
  return areaFromDevice;
}

thrust::host_vector<double> DPM2D::getPerimeters() {
  thrust::host_vector<double> perimeterFromDevice;
  perimeterFromDevice = d_perimeter;
  return perimeterFromDevice;
}

void DPM2D::calcParticlesShape() {
  // area and perimeter pointers
  const double *pos = thrust::raw_pointer_cast(&d_pos[0]);
  double *length = thrust::raw_pointer_cast(&d_length[0]);
  double *area = thrust::raw_pointer_cast(&d_area[0]);
  double *perimeter = thrust::raw_pointer_cast(&d_perimeter[0]);

  kernelCalcParticlesShape<<<dimGrid, dimBlock>>>(pos, length, area, perimeter);
}

void DPM2D::calcParticlesPositions() {
  // vertex and particle position pointers
  const double *pos = thrust::raw_pointer_cast(&d_pos[0]);
  double *particlePos = thrust::raw_pointer_cast(&d_particlePos[0]);

  kernelCalcParticlesPositions<<<dimGrid, dimBlock>>>(pos, particlePos);
}

void DPM2D::setDefaultParticleRadii() {
  for (long particleId = 0; particleId < numParticles; particleId++) {
    d_particleRad[particleId] = sqrt(d_a0[particleId]/PI);
  }
}

void DPM2D::setParticleRadii(thrust::host_vector<double> &particleRad_) {
  d_particleRad = particleRad_;
}

thrust::host_vector<double> DPM2D::getParticleRadii() {
  thrust::host_vector<double> particleRadFromDevice;
  particleRadFromDevice = d_particleRad;
  return particleRadFromDevice;
}

void DPM2D::setParticlePositions(thrust::host_vector<double> &particlePos_) {
  d_particlePos = particlePos_;
}

void DPM2D::setPBCParticlePositions(thrust::host_vector<double> &particlePos_) {
  d_particlePos = particlePos_;
  // check pbc
  thrust::device_vector<double> d_particlePosPBC(d_particlePos.size());
  const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
  double *pPosPBC = thrust::raw_pointer_cast(&d_particlePosPBC[0]);
  kernelCheckParticlePBC<<<partDimGrid, dimBlock>>>(pPosPBC, pPos);
  // copy to device
  d_particlePos = d_particlePosPBC;
}

thrust::host_vector<double> DPM2D::getParticlePositions() {
  thrust::host_vector<double> particlePosFromDevice;
  particlePosFromDevice = d_particlePos;
  return particlePosFromDevice;
}

thrust::host_vector<double> DPM2D::getPBCParticlePositions() {
  // check pbc
  thrust::device_vector<double> d_particlePosPBC(d_particlePos.size());
  const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
  double *pPosPBC = thrust::raw_pointer_cast(&d_particlePosPBC[0]);
  kernelCheckParticlePBC<<<partDimGrid, dimBlock>>>(pPosPBC, pPos);
  // copy to host
  thrust::host_vector<double> particlePosFromDevice;
  particlePosFromDevice = d_particlePosPBC;
  return particlePosFromDevice;
}

void DPM2D::resetPreviousPositions() {
  d_particlePreviousPos = getParticlePositions();
}

void DPM2D::resetLastPositions() {
  d_lastPos = getVertexPositions();
}

thrust::host_vector<double> DPM2D::getPreviousPositions() {
  thrust::host_vector<double> previousPosFromDevice;
  previousPosFromDevice = d_particlePreviousPos;
  return previousPosFromDevice;
}

void DPM2D::setParticleVelocities(thrust::host_vector<double> &particleVel_) {
  d_particleVel = particleVel_;
}

thrust::host_vector<double> DPM2D::getParticleVelocities() {
  thrust::host_vector<double> particleVelFromDevice;
  particleVelFromDevice = d_particleVel;
  return particleVelFromDevice;
}

void DPM2D::setParticleForces(thrust::host_vector<double> &particleForce_) {
  d_particleForce = particleForce_;
}

thrust::host_vector<double> DPM2D::getParticleForces() {
  thrust::host_vector<double> particleForceFromDevice;
  particleForceFromDevice = d_particleForce;
  return particleForceFromDevice;
}

thrust::host_vector<double> DPM2D::getParticleEnergies() {
  thrust::host_vector<double> particleEnergyFromDevice;
  particleEnergyFromDevice = d_particleEnergy;
  return particleEnergyFromDevice;
}

double DPM2D::getMeanParticleSize() {
  return sqrt(thrust::reduce(d_a0.begin(), d_a0.end(), double(0), thrust::plus<double>()) / (PI * numParticles));
}

double DPM2D::getMeanParticleSigma() {
  return thrust::reduce(d_particleRad.begin(), d_particleRad.end(), double(0), thrust::plus<double>()) / numParticles;
}

double DPM2D::getMinParticleSigma() {
  return thrust::reduce(d_particleRad.begin(), d_particleRad.end(), double(1), thrust::minimum<double>());
}

void DPM2D::setParticleAngles(thrust::host_vector<double> &particleAngle_) {
  d_particleAngle = particleAngle_;
}

thrust::host_vector<double> DPM2D::getParticleAngles() {
  thrust::host_vector<double> particleAngleFromDevice;
  particleAngleFromDevice = d_particleAngle;
  return particleAngleFromDevice;
}

//************************** dynamical variables *****************************//
void DPM2D::setVertexPositions(thrust::host_vector<double> &pos_) {
  d_pos = pos_;
}

thrust::host_vector<double> DPM2D::getVertexPositions() {
  thrust::host_vector<double> posFromDevice;
  posFromDevice = d_pos;
  return posFromDevice;
}

void DPM2D::setVertexVelocities(thrust::host_vector<double> &vel_) {
  d_vel = vel_;
}

thrust::host_vector<double> DPM2D::getVertexVelocities() {
  thrust::host_vector<double> velFromDevice;
  velFromDevice = d_vel;
  return velFromDevice;
}

void DPM2D::setVertexForces(thrust::host_vector<double> &force_) {
  d_force = force_;
}

thrust::host_vector<double> DPM2D::getVertexForces() {
  thrust::host_vector<double> forceFromDevice;
  forceFromDevice = d_force;
  return forceFromDevice;
}

void DPM2D::setVertexTorques(thrust::host_vector<double> &torque_) {
  d_torque = torque_;
}

thrust::host_vector<double> DPM2D::getVertexTorques() {
  thrust::host_vector<double> torqueFromDevice;
  torqueFromDevice = d_torque;
  return torqueFromDevice;
}

thrust::host_vector<double> DPM2D::getStressTensor() {
  calcStressTensor();
  thrust::host_vector<double> stressFromDevice;
  stressFromDevice = d_stress;
  return stressFromDevice;
}

thrust::host_vector<double> DPM2D::getPerParticleStressTensor() {
  calcPerParticleStressTensor();
  thrust::host_vector<double> perParticleStressFromDevice;
  perParticleStressFromDevice = d_perParticleStress;
  return perParticleStressFromDevice;
}

double DPM2D::getPressure() {
  calcStressTensor();
  double pressure = 0;
  for (long dim = 0; dim < nDim; dim++) {
    pressure += d_stress[dim * nDim + dim];
  }
  return pressure / (nDim * numVertices);
}

// return the sum of force magnitudes
double DPM2D::getTotalForceMagnitude() {
  thrust::device_vector<double> forceSquared(d_force.size());
  // compute squared velocities
  thrust::transform(d_force.begin(), d_force.end(), forceSquared.begin(), square());
  // sum squares
  double totalForceMagnitude = sqrt(thrust::reduce(forceSquared.begin(), forceSquared.end(), double(0), thrust::plus<double>()) / (numVertices * nDim));
  forceSquared.clear();
  return totalForceMagnitude;
}

// return the maximum force magnitude
double DPM2D::getMaxUnbalancedForce() {
  thrust::device_vector<double> forceSquared(d_force.size());
  // compute squared velocities
  thrust::transform(d_force.begin(), d_force.end(), forceSquared.begin(), square());

  double maxUnbalancedForce = sqrt(thrust::reduce(forceSquared.begin(), forceSquared.end(), double(-1), thrust::maximum<double>()));
  forceSquared.clear();
  return maxUnbalancedForce;
}

thrust::host_vector<long> DPM2D::getMaxNeighborList() {
  thrust::host_vector<long> maxNeighborListFromDevice;
  maxNeighborListFromDevice = d_maxNeighborList;
  return maxNeighborListFromDevice;
}

thrust::host_vector<long> DPM2D::getNeighbors() {
  thrust::host_vector<long> neighborListFromDevice;
  neighborListFromDevice = d_neighborList;
  return neighborListFromDevice;
}

thrust::host_vector<long> DPM2D::getContacts() {
  thrust::host_vector<long> contactListFromDevice;
  contactListFromDevice = d_contactList;
  return contactListFromDevice;
}

void DPM2D::printNeighbors() {
  for (long vertexId = 0; vertexId < numVertices; vertexId++) {
    cout << "vertexId: " << vertexId << " list of neighbors: ";
    for (long neighborId = 0; neighborId < d_maxNeighborList[vertexId]; neighborId++) {
      cout << d_neighborList[vertexId * neighborListSize + neighborId] << " ";
    }
    cout << endl;
  }
}

void DPM2D::printContacts() {
  for (long particleId = 0; particleId < numParticles; particleId++) {
    cout << "particleId: " << particleId << " list of contacts: ";
    for (long contactId = 0; contactId < d_numContacts[particleId]; contactId++) {
      cout << d_contactList[particleId * contactLimit + contactId] << " ";
    }
    cout << endl;
  }
}

double DPM2D::getPotentialEnergy() {
  return thrust::reduce(d_energy.begin(), d_energy.end(), double(0), thrust::plus<double>());
}

double DPM2D::getSmoothPotentialEnergy() {
  // the interaction energy is saved on the particle level and the shape energy
  // is saved on the vertex level for smooth interaction between vertices of different particles
  double totalEnergy = 0;
  totalEnergy = thrust::reduce(d_energy.begin(), d_energy.end(), double(0), thrust::plus<double>());
  totalEnergy += thrust::reduce(d_particleEnergy.begin(), d_particleEnergy.end(), double(0), thrust::plus<double>());
  return totalEnergy;
}

double DPM2D::getKineticEnergy() {
  thrust::device_vector<double> velSquared(d_vel.size());
  // compute squared velocities
  thrust::transform(d_vel.begin(), d_vel.end(), velSquared.begin(), square());
  // sum squares
  return 0.5 * thrust::reduce(velSquared.begin(), velSquared.end());
}

double DPM2D::getTemperature() {
  return 2. * getKineticEnergy() / (nDim * numVertices);
}

double DPM2D::getTotalEnergy() {
  double etot = getPotentialEnergy();
  etot += getKineticEnergy();
  return etot;
}

double DPM2D::getPhi() {
  double phi = double(thrust::reduce(d_area.begin(), d_area.end(), double(0), thrust::plus<double>()));
  // add effective vertex areas
  thrust::device_vector<double> d_vertexArea(d_area.size());
  double *vertexArea = thrust::raw_pointer_cast(&d_vertexArea[0]);
  const double *rad = thrust::raw_pointer_cast(&d_rad[0]);
  kernelCalcVertexArea<<<dimGrid,dimBlock>>>(rad, vertexArea);
  phi += PI * thrust::reduce(d_vertexArea.begin(), d_vertexArea.end(), double(0), thrust::plus<double>());
  return phi / (d_boxSize[0] * d_boxSize[1]);
}

double DPM2D::getPreferredPhi() {
  double phi = double(thrust::reduce(d_a0.begin(), d_a0.end(), double(0), thrust::plus<double>()));
  // add effective vertex areas
  thrust::device_vector<double> d_vertexArea(d_area.size());
  double *vertexArea = thrust::raw_pointer_cast(&d_vertexArea[0]);
  const double *rad = thrust::raw_pointer_cast(&d_rad[0]);
  kernelCalcVertexArea<<<dimGrid,dimBlock>>>(rad, vertexArea);
  phi += PI * thrust::reduce(d_vertexArea.begin(), d_vertexArea.end(), double(0), thrust::plus<double>());
  return phi / (d_boxSize[0] * d_boxSize[1]);
}

double DPM2D::getParticlePhi() {
  thrust::device_vector<double> d_radSquared(numParticles);
  thrust::transform(d_particleRad.begin(), d_particleRad.end(), d_radSquared.begin(), square());
  return thrust::reduce(d_radSquared.begin(), d_radSquared.end(), double(0), thrust::plus<double>()) * PI / (d_boxSize[0] * d_boxSize[1]);
}

double DPM2D::get3DParticlePhi() {
  thrust::device_vector<double> d_volume(numParticles);
  thrust::fill(d_volume.begin(), d_volume.end(), double(1));
  long p_nDim(nDim);
  auto r = thrust::counting_iterator<long>(0);
  double *volume = thrust::raw_pointer_cast(&d_volume[0]);
  const double *rad = thrust::raw_pointer_cast(&d_particleRad[0]);

  auto computeVolume = [=] __device__ (long particleId) {
    #pragma unroll (MAXDIM)
		for (long dim = 0; dim < p_nDim; dim++) {
      volume[particleId] *= rad[particleId];
    }
  };

  thrust::for_each(r, r + numParticles, computeVolume);
  return thrust::reduce(d_volume.begin(), d_volume.end(), double(0), thrust::plus<double>()) * 3 * PI / (4 * d_boxSize[0] * d_boxSize[1] * d_boxSize[2]);
}

double DPM2D::getVertexMSD() {
  const double *pos = thrust::raw_pointer_cast(&d_pos[0]);
  const double *initialPos = thrust::raw_pointer_cast(&d_initialPos[0]);
  double *delta = thrust::raw_pointer_cast(&d_delta[0]);
  kernelCalcVertexDistanceSq<<<dimGrid,dimBlock>>>(pos, initialPos, delta);
  return thrust::reduce(d_delta.begin(), d_delta.end(), double(0), thrust::plus<double>()) / (numVertices * d_boxSize[0] * d_boxSize[1]);
}

double DPM2D::getMaxDisplacement() {
  const double *pos = thrust::raw_pointer_cast(&d_pos[0]);
  const double *lastPos = thrust::raw_pointer_cast(&d_lastPos[0]);
  double *disp = thrust::raw_pointer_cast(&d_disp[0]);
  kernelCalcVertexDisplacement<<<dimGrid,dimBlock>>>(pos, lastPos, disp);
  return thrust::reduce(d_disp.begin(), d_disp.end(), double(-1), thrust::maximum<double>());
}

double DPM2D::getParticleMSD() {
  const double *particlePos = thrust::raw_pointer_cast(&d_particlePos[0]);
  const double *particleInitPos = thrust::raw_pointer_cast(&d_particleInitPos[0]);
  double *particleDelta = thrust::raw_pointer_cast(&d_particleDelta[0]);
  kernelCalcParticleDistanceSq<<<partDimGrid,dimBlock>>>(particlePos, particleInitPos, particleDelta);
  return thrust::reduce(d_particleDelta.begin(), d_particleDelta.end(), double(0), thrust::plus<double>()) / (numParticles * d_boxSize[0] * d_boxSize[1]);
}

double DPM2D::getParticleMaxDisplacement() {
  const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
  const double *pPrevPos = thrust::raw_pointer_cast(&d_particlePreviousPos[0]);
  double *pDisp = thrust::raw_pointer_cast(&d_particleDisp[0]);
  kernelCalcParticleDisplacement<<<partDimGrid,dimBlock>>>(pPos, pPrevPos, pDisp);
  return thrust::reduce(d_particleDisp.begin(), d_particleDisp.end(), double(-1), thrust::maximum<double>());
  //auto r = thrust::counting_iterator<long>(0);
  //const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
  //const double *pPrevPos = thrust::raw_pointer_cast(&d_particlePreviousPos[0]);

	//auto perParticleDistance = [=] __device__ (int i) {
	//	return calcDistance(&pPos[i*d_nDim], &pPrevPos[i*d_nDim]);
	//};

	//return double( thrust::transform_reduce( r, r + numParticles, perParticleDistance, double(-1), thrust::maximum<double>()) );
}

double DPM2D::getDeformableWaveNumber() {
  return PI / (2. * sqrt(d_boxSize[0] * d_boxSize[1] * getPhi() / (PI * numParticles)));
}

double DPM2D::getSoftWaveNumber() {
  if(nDim == 2) {
    return PI / (2. * sqrt(d_boxSize[0] * d_boxSize[1] * getParticlePhi() / (PI * numParticles)));
  } else if(nDim == 3) {
    return PI / (2. * sqrt(d_boxSize[0] * d_boxSize[1] * get3DParticlePhi() / (PI * numParticles)));
  } else {
    cout << "DPM2D::getSoftWaveNumber: this function works only for dim = 2 and 3" << endl;
    return 0;
  }
}

double DPM2D::getVertexISF() {
  double vertexWaveNumber = PI / (2 * d_rad[0]);
  thrust::device_vector<double> d_vertexSF(numVertices);
  const double *pos = thrust::raw_pointer_cast(&d_pos[0]);
  const double *initialPos = thrust::raw_pointer_cast(&d_initialPos[0]);
  double *vertexSF = thrust::raw_pointer_cast(&d_vertexSF[0]);
  kernelCalcVertexScatteringFunction<<<dimGrid,dimBlock>>>(pos, initialPos, vertexSF, vertexWaveNumber);
  return thrust::reduce(d_vertexSF.begin(), d_vertexSF.end(), double(0), thrust::plus<double>()) / numVertices;
}

double DPM2D::getParticleISF(double waveNumber_) {
  thrust::device_vector<double> d_particleSF(numParticles);
  const double *particlePos = thrust::raw_pointer_cast(&d_particlePos[0]);
  const double *particleInitPos = thrust::raw_pointer_cast(&d_particleInitPos[0]);
  double *particleSF = thrust::raw_pointer_cast(&d_particleSF[0]);
  kernelCalcParticleScatteringFunction<<<partDimGrid,dimBlock>>>(particlePos, particleInitPos, particleSF, waveNumber_);
  return thrust::reduce(d_particleSF.begin(), d_particleSF.end(), double(0), thrust::plus<double>()) / numParticles;
}

double DPM2D::getHexaticOrderParameter() {
  thrust::device_vector<double> d_psi6(numParticles);
  thrust::fill(d_psi6.begin(), d_psi6.end(), double(0));
  double *psi6 = thrust::raw_pointer_cast(&d_psi6[0]);
  const double *particlePos = thrust::raw_pointer_cast(&d_particlePos[0]);
  kernelCalcHexaticOrderParameter<<<dimGrid,dimBlock>>>(particlePos, psi6);
  return thrust::reduce(d_psi6.begin(), d_psi6.end(), double(0), thrust::plus<double>()) / numParticles;
}

double DPM2D::getAreaFluctuation() {
  thrust::device_vector<double> deltaA(d_area.size());
  thrust::device_vector<double> deltaASq(d_area.size());
  thrust::fill(deltaA.begin(), deltaA.end(), double(0));
  thrust::transform(d_area.begin(), d_area.end(), d_a0.begin(), deltaA.begin(), thrust::minus<double>());
  thrust::transform(deltaA.begin(), deltaA.end(), deltaASq.begin(), square());
  return sqrt(thrust::reduce(deltaASq.begin(), deltaASq.end(), double(0), thrust::plus<double>()) / numParticles);
}

//************************ initilization functions ***************************//
void DPM2D::setMonoSizeDistribution() {
  thrust::fill(d_numVertexInParticleList.begin(), d_numVertexInParticleList.end(), numVertexPerParticle);
  long* numVertexInParticleList = thrust::raw_pointer_cast(&d_numVertexInParticleList[0]);
  hipMemcpyToSymbol(HIP_SYMBOL(d_numVertexInParticleListPtr), &numVertexInParticleList, sizeof(numVertexInParticleList));
}

//void DPM2D::setBiSizeDistribution();

void DPM2D::setPolyRandomSoftParticles(double phi0, double polyDispersity) {
  double r1, r2, randNum, mean, sigma, scale, boxLength = 1.;
  mean = 0.;
  sigma = sqrt(log(polyDispersity*polyDispersity + 1.));
  // generate polydisperse particle size
  for (long particleId = 0; particleId < numParticles; particleId++) {
    r1 = drand48();
    r2 = drand48();
    randNum = sqrt(-2. * log(r1)) * cos(2. * PI * r2);
    d_particleRad[particleId] = exp(mean + randNum * sigma);
    d_a0[particleId] = PI * d_particleRad[particleId] * d_particleRad[particleId];
  }
  scale = sqrt(getParticlePhi() / phi0); // sqrt for 2d
  for (long dim = 0; dim < nDim; dim++) {
    d_boxSize[dim] = boxLength;
  }
  double* boxSize = thrust::raw_pointer_cast(&(d_boxSize[0]));
  hipMemcpyToSymbol(HIP_SYMBOL(d_boxSizePtr), &boxSize, sizeof(boxSize));
  // extract random positions
  double areaSum = 0;
  for (long particleId = 0; particleId < numParticles; particleId++) {
    d_particleRad[particleId] /= scale;
    d_a0[particleId] = PI * d_particleRad[particleId] * d_particleRad[particleId];
    for(long dim = 0; dim < nDim; dim++) {
      d_particlePos[particleId * nDim + dim] = d_boxSize[dim] * drand48();
    }
    areaSum += d_a0[particleId];
  }
  // need to set this otherwise forces are zeros
  setLengthScaleToOne();
  //setSphericalLengthScale();
  cout << "DPM2D::setPolyRandomSoftParticles: particle packing fraction: " << getParticlePhi() << endl;
}

void DPM2D::setPolySizeDistribution(double calA0_, double polyDispersity) {
  calA0 = calA0_;
  double r1, r2, randNum, calA0temp;
  double numVertexInParticle, minVertexInParticle = numVertexPerParticle; // default
  numVertices = 0;
  // generate polydisperse number of vertices per particle
  for (long particleId = 0; particleId < numParticles; particleId++) {
    r1 = drand48();
    r2 = drand48();
    randNum = sqrt(-2. * log(r1)) * cos(2. * PI * r2);
    numVertexInParticle = floor(polyDispersity * numVertexPerParticle * randNum + numVertexPerParticle);
    if(numVertexInParticle < minVertexInParticle) {
      numVertexInParticle = minVertexInParticle;
    }
    // each particle has at least numVertexPerParticle vertices
    d_numVertexInParticleList[particleId] = numVertexInParticle;
    numVertices += numVertexInParticle;
  }
  cout << "DPM2D::setPolySizeDistribution: numVertices: " << numVertices << endl;
  hipMemcpyToSymbol(HIP_SYMBOL(d_numVertices), &(numVertices), sizeof(numVertices));
  setDimBlock(dimBlock); // recalculate dimGrid
  long* numVertexInParticleList = thrust::raw_pointer_cast(&d_numVertexInParticleList[0]);
  hipMemcpyToSymbol(HIP_SYMBOL(d_numVertexInParticleListPtr), &numVertexInParticleList, sizeof(numVertexInParticleList));

  // initialize everything else
  initParticleIdList();
  // we changed numVertices so we need to resize variables
  initVertexVariables(numVertices);
  initDynamicalVariables(numVertices);
  initNeighbors(numVertices);
  for (long particleId = 0; particleId < numParticles; particleId++) {
    numVertexInParticle = d_numVertexInParticleList[particleId];
    d_a0[particleId] = (numVertexInParticle / minVertexInParticle) * (numVertexInParticle / minVertexInParticle);
    calA0temp = calA0 * numVertexInParticle * tan(PI / numVertexInParticle) / PI;
    for (long vertexId = 0; vertexId < numVertexInParticle; vertexId++) {
      d_l0[d_firstVertexInParticleId[particleId] + vertexId] = 2. * sqrt(PI * calA0temp * d_a0[particleId]) / numVertexInParticle;
  		d_theta0[d_firstVertexInParticleId[particleId] + vertexId] = 2. * PI / numVertexInParticle;
  		d_rad[d_firstVertexInParticleId[particleId] + vertexId] = 0.5 * d_l0[d_firstVertexInParticleId[particleId] + vertexId];
      //cout << "vertexId: " << d_firstVertexInParticleId[particleId] + vertexId << " l0: " << d_l0[d_firstVertexInParticleId[particleId] + vertexId] << " rad: " << d_rad[d_firstVertexInParticleId[particleId] + vertexId] << endl;
    }
  }
}

void DPM2D::setSinusoidalRestAngles(double thetaA, double thetaK) {
  double thetaR;
  for (long particleId = 0; particleId < numParticles; particleId++) {
    thetaR = 2. * PI / d_numVertexInParticleList[particleId];
    for (long vertexId = 0; vertexId < d_numVertexInParticleList[particleId]; vertexId++) {
      d_theta0[d_firstVertexInParticleId[particleId] + vertexId] = thetaA * thetaR * cos(thetaR * thetaK * vertexId);
    }
  }
}

// this works only for a square box
void DPM2D::setRandomParticles(double phi0, double extraRad_) {
  double boxLength = 1., scale = sqrt(getPreferredPhi() / phi0), extraRad = extraRad_;
  for (long dim = 0; dim < nDim; dim++) {
    d_boxSize[dim] = boxLength; // sqrt for 2d
  }
  double* boxSize = thrust::raw_pointer_cast(&(d_boxSize[0]));
  hipMemcpyToSymbol(HIP_SYMBOL(d_boxSizePtr), &boxSize, sizeof(boxSize));
  // extract random positions and radii
  double areaSum = 0;
  for (long particleId = 0; particleId < numParticles; particleId++) {
    d_a0[particleId] /= (scale * scale);
    for(long dim = 0; dim < nDim; dim++) {
      d_particlePos[particleId * nDim + dim] = d_boxSize[dim] * drand48();
    }
    d_particleRad[particleId] = extraRad * sqrt((2. * d_a0[particleId]) / (d_numVertexInParticleList[particleId] * sin(2. * PI / d_numVertexInParticleList[particleId])));
    areaSum += PI * d_particleRad[particleId] * d_particleRad[particleId];
  }
  for(long vertexId = 0; vertexId < numVertices; vertexId++) {
    d_l0[vertexId] /= scale;
    d_rad[vertexId] /= scale;
  }
  // need to set this otherwise forces are zeros
  setLengthScale();
  cout << "DPM2D::setRandomParticles: particle packing fraction: " << getPreferredPhi() << " " << areaSum/(boxLength*boxLength) << endl;
}

void DPM2D::initVerticesOnParticles() {
  double rad;
  long particleId, numVertexInParticle;
  for (long vertexId = 0; vertexId < numVertices; vertexId++) {
    particleId = d_particleIdList[vertexId];
    numVertexInParticle = d_numVertexInParticleList[particleId];
    rad = sqrt((2. * d_a0[particleId]) / (numVertexInParticle * sin(2. * PI / numVertexInParticle)));
		d_pos[vertexId * nDim] = rad * cos((2. * PI * vertexId) / numVertexInParticle) + d_particlePos[particleId * nDim] + 1e-02 * d_l0[vertexId] * drand48();
		d_pos[vertexId * nDim + 1] = rad * sin((2. * PI * vertexId) / numVertexInParticle) + d_particlePos[particleId * nDim + 1] + 1e-02 * d_l0[vertexId] * drand48();
  }
}

void DPM2D::scaleVertices(double scale) {
  thrust::host_vector<double> distance(nDim);
  calcParticlesPositions();
  for (long vertexId = 0; vertexId < numVertices; vertexId++) {
    for (long dim = 0; dim < nDim; dim++) {
      distance[dim] = d_pos[vertexId * nDim + dim] - d_particlePos[d_particleIdList[vertexId]];
      d_pos[vertexId * nDim + dim] += (scale - 1.) * distance[dim];
    }
  }
  thrust::transform(d_a0.begin(), d_a0.end(), thrust::make_constant_iterator(scale * scale), d_a0.begin(), thrust::multiplies<double>());
  thrust::transform(d_area.begin(), d_area.end(), thrust::make_constant_iterator(scale * scale), d_area.begin(), thrust::multiplies<double>());
  thrust::transform(d_l0.begin(), d_l0.end(), thrust::make_constant_iterator(scale), d_l0.begin(), thrust::multiplies<double>());
  thrust::transform(d_rad.begin(), d_rad.end(), thrust::make_constant_iterator(scale), d_rad.begin(), thrust::multiplies<double>());
  distance.clear();
  setLengthScale();
}

void DPM2D::scaleParticles(double scale) {
  thrust::transform(d_particleRad.begin(), d_particleRad.end(), thrust::make_constant_iterator(scale), d_particleRad.begin(), thrust::multiplies<double>());
  thrust::transform(d_a0.begin(), d_a0.end(), thrust::make_constant_iterator(scale * scale), d_a0.begin(), thrust::multiplies<double>());
  setLengthScale();
}

void DPM2D::pressureScaleParticles(double pscale) {
  thrust::transform(d_particlePos.begin(), d_particlePos.end(), thrust::make_constant_iterator(pscale), d_particlePos.begin(), thrust::multiplies<double>());
  thrust::transform(d_boxSize.begin(), d_boxSize.end(), thrust::make_constant_iterator(pscale), d_boxSize.begin(), thrust::multiplies<double>());
}

void DPM2D::scaleSoftParticles(double scale) {
  thrust::transform(d_particleRad.begin(), d_particleRad.end(), thrust::make_constant_iterator(scale), d_particleRad.begin(), thrust::multiplies<double>());
  thrust::transform(d_a0.begin(), d_a0.end(), thrust::make_constant_iterator(scale * scale), d_a0.begin(), thrust::multiplies<double>());
  //setSphericalLengthScale();
}

void DPM2D::scaleParticleVelocity(double scale) {
  thrust::transform(d_particleVel.begin(), d_particleVel.end(), thrust::make_constant_iterator(scale), d_particleVel.begin(), thrust::multiplies<double>());
}

// translate vertices by particle displacement
void DPM2D::translateVertices() {
  thrust::transform(d_particlePos.begin(), d_particlePos.end(), d_particleInitPos.begin(), d_particleDelta.begin(), thrust::minus<double>());
	double* pos = thrust::raw_pointer_cast(&d_pos[0]);
	const double* pDelta = thrust::raw_pointer_cast(&d_particleDelta[0]);
  kernelTranslateVertices<<<dimGrid, dimBlock>>>(pDelta, pos);
}

// rotate vertices by particle angle change
void DPM2D::rotateVertices() {
	thrust::transform(d_particleAngle.begin(), d_particleAngle.end(), d_particleInitAngle.begin(), d_particleDeltaAngle.begin(), thrust::minus<double>());
	double* pos = thrust::raw_pointer_cast(&d_pos[0]);
	const double* particlePos = thrust::raw_pointer_cast(&d_particlePos[0]);
	const double* pDeltaAngle = thrust::raw_pointer_cast(&d_particleDeltaAngle[0]);
  kernelRotateVertices<<<dimGrid, dimBlock>>>(pDeltaAngle, particlePos, pos);
}

// compute particle angles from velocity
void DPM2D::computeParticleAngleFromVel() {
  long p_nDim(nDim);
  auto r = thrust::counting_iterator<long>(0);
  double* pAngle = thrust::raw_pointer_cast(&d_particleAngle[0]);
  const double* pVel = thrust::raw_pointer_cast(&d_particleVel[0]);

  auto computeParticleAngle = [=] __device__ (long particleId) {
    pAngle[particleId] = atan(pVel[particleId * p_nDim + 1] / pVel[particleId * p_nDim]);
  };

  thrust::for_each(r, r + numParticles, computeParticleAngle);
}

//*************************** force and energy *******************************//
void DPM2D::setEnergyCosts(double ea_, double el_, double eb_, double ec_) {
  ea = ea_;
  el = el_;
  eb = eb_;
  ec = ec_;
  hipMemcpyToSymbol(HIP_SYMBOL(d_ea), &ea, sizeof(ea));
  hipMemcpyToSymbol(HIP_SYMBOL(d_el), &el, sizeof(el));
  hipMemcpyToSymbol(HIP_SYMBOL(d_eb), &eb, sizeof(eb));
  hipMemcpyToSymbol(HIP_SYMBOL(d_ec), &ec, sizeof(ec));
}

void DPM2D::setAttractionConstants(double l1_, double l2_) {
  l1 = l1_;
  l2 = l2_;
  hipMemcpyToSymbol(HIP_SYMBOL(d_l1), &l1, sizeof(l1));
  hipMemcpyToSymbol(HIP_SYMBOL(d_l2), &l2, sizeof(l2));
}

void DPM2D::setLJcutoff(double LJcutoff_) {
  LJcutoff = LJcutoff_;
  hipMemcpyToSymbol(HIP_SYMBOL(d_LJcutoff), &LJcutoff, sizeof(LJcutoff));
  LJecut = 4 * (1 / pow(LJcutoff, 12) - 1 / pow(LJcutoff, 6));
  hipMemcpyToSymbol(HIP_SYMBOL(d_LJecut), &LJecut, sizeof(LJecut));
  //cout << "DPM2D::setLJcutoff - LJcutoff: " << LJcutoff << " LJecut: " << LJecut << endl;
}

double DPM2D::setTimeScale(double dt_) {
  double ta, tl, tb, tmin = 1e08;
  // compute typical time scale
  ta = rho0 / sqrt(ea);
  tl = (rho0 * d_l0[0]) / sqrt(ea * el); // TODO: replace values at 0 with averages
  tb = (rho0 * d_l0[0]) / sqrt(ea * eb); // TODO: replace values at 0 with averages
  // compute global time scale
  if (ta < tmin) tmin = ta;
  if (tl < tmin) tmin = tl;
  if (tb < tmin) tmin = tb;
  dt = tmin * dt_;
  hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &dt, sizeof(dt));
  return dt;
}

double DPM2D::setTimeStep(double dt_) {
  dt = dt_;
  hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &dt, sizeof(dt));
  return dt;
}

void DPM2D::calcForceEnergy() {
  thrust::fill(d_energy.begin(), d_energy.end(), double(0));
  calcParticlesShape();
  calcParticlesPositions();
  // shape variables
	const double *a0 = thrust::raw_pointer_cast(&d_a0[0]);
	const double *l0 = thrust::raw_pointer_cast(&d_l0[0]);
  const double *rad = thrust::raw_pointer_cast(&d_rad[0]);
	const double *theta0 = thrust::raw_pointer_cast(&d_theta0[0]);
  // dynamical variables
  const double *area = thrust::raw_pointer_cast(&d_area[0]);
  const double *particlePos = thrust::raw_pointer_cast(&d_particlePos[0]);
  const double *pos = thrust::raw_pointer_cast(&d_pos[0]);
	double *force = thrust::raw_pointer_cast(&d_force[0]);
	double *energy = thrust::raw_pointer_cast(&d_energy[0]);
  // compute shape force and vertex interaction
  kernelCalcShapeForceEnergy<<<dimGrid, dimBlock>>>(a0, area, particlePos, l0, theta0, pos, force, energy);
  //kernelCalcVertexInteraction<<<dimGrid, dimBlock>>>(rad, pos, force, energy);
  thrust::fill(d_particleEnergy.begin(), d_particleEnergy.end(), double(0));
  double *pEnergy = thrust::raw_pointer_cast(&d_particleEnergy[0]);
  kernelCalcVertexSegmentInteraction<<<dimGrid, dimBlock>>>(rad, pos, force, pEnergy);
}

void DPM2D::calcVertexForceAngAcc() {
  calcParticlesPositions();
  const double *rad = thrust::raw_pointer_cast(&d_rad[0]);
  const double *pos = thrust::raw_pointer_cast(&d_pos[0]);
  const double *particlePos = thrust::raw_pointer_cast(&d_particlePos[0]);
  double *force = thrust::raw_pointer_cast(&d_force[0]);
  double *torque = thrust::raw_pointer_cast(&d_torque[0]);
	double *energy = thrust::raw_pointer_cast(&d_energy[0]);
  // torque here is used for angular acceleration
  kernelCalcVertexForceAngAcc<<<dimGrid, dimBlock>>>(rad, pos, particlePos, force, torque, energy);
}

void DPM2D::calcRigidForceEnergy() {
  calcVertexForceAngAcc();
  // vertex variables
	const double *force = thrust::raw_pointer_cast(&d_force[0]);
  const double *torque = thrust::raw_pointer_cast(&d_torque[0]);
	const double *energy = thrust::raw_pointer_cast(&d_energy[0]);
  // particle variables
	double *pForce = thrust::raw_pointer_cast(&d_particleForce[0]);
  double *pTorque = thrust::raw_pointer_cast(&d_particleTorque[0]);
  double *pEnergy = thrust::raw_pointer_cast(&d_particleEnergy[0]);
  // sum force and torque over vertices of particle
  kernelCalcParticleForceAngAcc<<<dimGrid, dimBlock>>>(force, torque, energy, pForce, pTorque, pEnergy);
}

void DPM2D::calcVertexForceTorque() {
  calcParticlesPositions();
  const double *rad = thrust::raw_pointer_cast(&d_rad[0]);
  const double *pos = thrust::raw_pointer_cast(&d_pos[0]);
  const double *particlePos = thrust::raw_pointer_cast(&d_particlePos[0]);
  double *force = thrust::raw_pointer_cast(&d_force[0]);
  double *torque = thrust::raw_pointer_cast(&d_torque[0]);
	double *energy = thrust::raw_pointer_cast(&d_energy[0]);
  // torque here is used for angular acceleration
  kernelCalcVertexForceTorque<<<dimGrid, dimBlock>>>(rad, pos, particlePos, force, torque, energy);
}

void DPM2D::calcRigidForceTorque() {
  calcVertexForceTorque();
  // vertex variables
	const double *force = thrust::raw_pointer_cast(&d_force[0]);
  const double *torque = thrust::raw_pointer_cast(&d_torque[0]);
  // particle variables
	double *pForce = thrust::raw_pointer_cast(&d_particleForce[0]);
  double *pTorque = thrust::raw_pointer_cast(&d_particleTorque[0]);
  // sum force and torque over vertices of particle
  kernelCalcParticleForceTorque<<<dimGrid, dimBlock>>>(force, torque, pForce, pTorque);
}

void DPM2D::calcStressTensor() {
  calcPerParticleStressTensor();
  thrust::fill(d_stress.begin(), d_stress.end(), double(0));
  const double *perPStress = thrust::raw_pointer_cast(&d_perParticleStress[0]);
	double *stress = thrust::raw_pointer_cast(&d_stress[0]);
  kernelCalcStressTensor<<<partDimGrid, dimBlock>>>(perPStress, stress);
}

void DPM2D::calcPerParticleStressTensor() {
  thrust::fill(d_stress.begin(), d_stress.end(), double(0));
  const double *rad = thrust::raw_pointer_cast(&d_rad[0]);
	const double *pos = thrust::raw_pointer_cast(&d_pos[0]);
	const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
	double *perPStress = thrust::raw_pointer_cast(&d_perParticleStress[0]);
  kernelCalcPerParticleStressTensor<<<partDimGrid, dimBlock>>>(rad, pos, pPos, perPStress);
}

void DPM2D::calcNeighborForces() {
  thrust::host_vector<double> neighborForce;
  neighborForce.resize(numVertices * neighborListSize * nDim);
  thrust::fill(neighborForce.begin(), neighborForce.end(), 0);
	const double *rad = thrust::raw_pointer_cast(&d_rad[0]);
  const double *pos = thrust::raw_pointer_cast(&d_pos[0]);
  double *neighforce = thrust::raw_pointer_cast(&neighborForce[0]);
  kernelCalcNeighborForces<<<dimGrid, dimBlock>>>(pos, rad, neighforce);
}

//************************* contacts and neighbors ***************************//
void DPM2D::calcParticleNeighbors() {
  long largestNeighbor = 8*nDim; // Guess
	do {
		//Make a contactList that is the right size
		neighborLimit = largestNeighbor;
		d_partNeighborList = thrust::device_vector<long>(numParticles * neighborLimit);
		//Prefill the contactList with -1
		thrust::fill(d_partNeighborList.begin(), d_partNeighborList.end(), -1L);
		thrust::fill(d_numPartNeighbors.begin(), d_numPartNeighbors.end(), -1L);
		//Create device_pointers from thrust arrays
		double* pos = thrust::raw_pointer_cast(&d_pos[0]);
		double* rad = thrust::raw_pointer_cast(&d_rad[0]);
		long* pNeighborList = thrust::raw_pointer_cast(&d_partNeighborList[0]);
		long* numPNeighbors = thrust::raw_pointer_cast(&d_numPartNeighbors[0]);
		kernelCalcParticleNeighbors<<<dimGrid, dimBlock>>>(pos, rad, neighborLimit, pNeighborList, numPNeighbors);
		//Calculate the maximum number of contacts
		largestNeighbor = thrust::reduce(d_numPartNeighbors.begin(), d_numPartNeighbors.end(), -1L, thrust::maximum<long>());
    //cout << "DPM2D::calcParticleNeighbors: largestNeighbor = " << largestNeighbor << endl;
	} while(neighborLimit < largestNeighbor); // If the guess was not good, do it again
}

void DPM2D::calcContacts(double gapSize) {
  long largestContact = 8*nDim; // Guess
	do {
		//Make a contactList that is the right size
		contactLimit = largestContact;
		d_contactList = thrust::device_vector<long>(numParticles * contactLimit);
		//Prefill the contactList with -1
		thrust::fill(d_contactList.begin(), d_contactList.end(), -1L);
		thrust::fill(d_numContacts.begin(), d_numContacts.end(), -1L);
		//Create device_pointers from thrust arrays
		const double* pos = thrust::raw_pointer_cast(&d_pos[0]);
		const double* rad = thrust::raw_pointer_cast(&d_rad[0]);
		long* contactList = thrust::raw_pointer_cast(&d_contactList[0]);
		long* numContacts = thrust::raw_pointer_cast(&d_numContacts[0]);
		kernelCalcContacts<<<dimGrid, dimBlock>>>(pos, rad, gapSize, contactLimit, contactList, numContacts);
		//Calculate the maximum number of contacts
		largestContact = thrust::reduce(d_numContacts.begin(), d_numContacts.end(), -1L, thrust::maximum<long>());
    //cout << "DPM2D::calcContacts: largestContact = " << largestContact << endl;
	} while(contactLimit < largestContact); // If the guess was not good, do it again
}

//Return normalized contact vectors between every pair of particles in contact
thrust::host_vector<long> DPM2D::getContactVectors(double gapSize) {
	//Calculate the set of contacts
	calcContacts(gapSize);
	//Calculate the maximum number of contacts
	maxContacts = thrust::reduce(d_numContacts.begin(), d_numContacts.end(), -1L, thrust::maximum<long>());
	//Create the array to hold the contactVectors
	d_contactVectorList.resize(numParticles * nDim * maxContacts);
	thrust::fill(d_contactVectorList.begin(), d_contactVectorList.end(), double(0));
	double* pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
	long* contactList = thrust::raw_pointer_cast(&d_contactList[0]);
	double* contactVectorList = thrust::raw_pointer_cast(&d_contactVectorList[0]);
	kernelCalcContactVectorList<<<dimGrid, dimBlock>>>(pPos, contactList, d_contactList.size()/numParticles, maxContacts, contactVectorList);
  // convert to host and return
  thrust::host_vector<long> contactVectorListFromDevice;
  contactVectorListFromDevice = d_contactVectorList;
  return contactVectorListFromDevice;
}

//*************************** vertex neighbors *******************************//
void DPM2D::calcNeighborList(double cutDistance) {
  thrust::fill(d_maxNeighborList.begin(), d_maxNeighborList.end(), 0);
	thrust::fill(d_neighborList.begin(), d_neighborList.end(), -1L);
  syncNeighborsToDevice();

  const double *pos = thrust::raw_pointer_cast(&d_pos[0]);
	const double *rad = thrust::raw_pointer_cast(&d_rad[0]);

  kernelCalcNeighborList<<<dimGrid, dimBlock>>>(pos, rad, cutDistance);
  // compute maximum number of neighbors per particle
  maxNeighbors = thrust::reduce(d_maxNeighborList.begin(), d_maxNeighborList.end(), -1L, thrust::maximum<long>());
  syncNeighborsToDevice();
  //cout << "\n DPM2D::calcNeighborList: maxNeighbors = " << maxNeighbors << endl;

  // if the neighbors don't fit, resize the neighbor list
  if ( maxNeighbors > neighborListSize ) {
		neighborListSize = pow(2, ceil(std::log2(maxNeighbors)));
    //cout << "neighborListSize: " << neighborListSize << endl;
		//Now create the actual storage and then put the neighbors in it.
		d_neighborList.resize(numVertices * neighborListSize);
		//Pre-fill the neighborList with -1
		thrust::fill(d_neighborList.begin(), d_neighborList.end(), -1L);
		syncNeighborsToDevice();
		kernelCalcNeighborList<<<dimGrid, dimBlock>>>(pos, rad, cutDistance);
	}
}

void DPM2D::syncNeighborsToDevice() {
	//Copy the pointers and information about neighbors to the gpu
	hipMemcpyToSymbol(HIP_SYMBOL(d_neighborListSize), &neighborListSize, sizeof(neighborListSize));
	hipMemcpyToSymbol(HIP_SYMBOL(d_maxNeighbors), &maxNeighbors, sizeof(maxNeighbors));

	long* maxNeighborList = thrust::raw_pointer_cast(&d_maxNeighborList[0]);
	hipMemcpyToSymbol(HIP_SYMBOL(d_maxNeighborListPtr), &maxNeighborList, sizeof(maxNeighborList));

	long* neighborList = thrust::raw_pointer_cast(&d_neighborList[0]);
	hipMemcpyToSymbol(HIP_SYMBOL(d_neighborListPtr), &neighborList, sizeof(neighborList));
}

//************************* particle functions *******************************//
void DPM2D::calcParticleForceEnergy() {
	const double *pRad = thrust::raw_pointer_cast(&d_particleRad[0]);
	const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
	double *pForce = thrust::raw_pointer_cast(&d_particleForce[0]);
	double *pEnergy = thrust::raw_pointer_cast(&d_particleEnergy[0]);
  // compute particle interaction
  kernelCalcParticleInteraction<<<partDimGrid, dimBlock>>>(pRad, pPos, pForce, pEnergy);
}

void DPM2D::calcParticleWallForceEnergy() {
  const double *pRad = thrust::raw_pointer_cast(&d_particleRad[0]);
  const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
  double *pForce = thrust::raw_pointer_cast(&d_particleForce[0]);
  double *pEnergy = thrust::raw_pointer_cast(&d_particleEnergy[0]);
  // compute particle interaction
  kernelCalcParticleInteractionFixedBoundary<<<partDimGrid, dimBlock>>>(pRad, pPos, pForce, pEnergy);
  kernelCalcParticleWallInteraction<<<partDimGrid, dimBlock>>>(pRad, pPos, pForce, pEnergy);
}

void DPM2D::calcParticleSidesForceEnergy() {
  const double *pRad = thrust::raw_pointer_cast(&d_particleRad[0]);
  const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
  double *pForce = thrust::raw_pointer_cast(&d_particleForce[0]);
  double *pEnergy = thrust::raw_pointer_cast(&d_particleEnergy[0]);
  // compute particle interaction
  kernelCalcParticleInteractionFixedSides<<<partDimGrid, dimBlock>>>(pRad, pPos, pForce, pEnergy);
  kernelCalcParticleSidesInteraction<<<partDimGrid, dimBlock>>>(pRad, pPos, pForce, pEnergy);
}

void DPM2D::calcParticleForceEnergyRA() { // Repulsive and Attractive
	const double *pRad = thrust::raw_pointer_cast(&d_particleRad[0]);
	const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
	double *pForce = thrust::raw_pointer_cast(&d_particleForce[0]);
	double *pEnergy = thrust::raw_pointer_cast(&d_particleEnergy[0]);
  // compute particle interaction
  kernelCalcParticleInteractionRA<<<partDimGrid, dimBlock>>>(pRad, pPos, pForce, pEnergy);
}

void DPM2D::calcParticleForceEnergyLJ() { // Repulsive and Attractive
	const double *pRad = thrust::raw_pointer_cast(&d_particleRad[0]);
	const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
	double *pForce = thrust::raw_pointer_cast(&d_particleForce[0]);
	double *pEnergy = thrust::raw_pointer_cast(&d_particleEnergy[0]);
  // compute particle interaction
  kernelCalcParticleInteractionLJ<<<partDimGrid, dimBlock>>>(pRad, pPos, pForce, pEnergy);
}

 void DPM2D::makeExternalParticleForce(double externalForce) {
   // extract +-1 random forces
   d_particleDelta.resize(numParticles);
   thrust::fill(d_particleDelta.begin(), d_particleDelta.end(), double(0));
   thrust::counting_iterator<long> index_sequence_begin(lrand48());
   thrust::transform(index_sequence_begin, index_sequence_begin + numParticles, d_particleDelta.begin(), randInt(0,1));
   thrust::transform(d_particleDelta.begin(), d_particleDelta.end(), thrust::make_constant_iterator(2), d_particleDelta.begin(), thrust::multiplies<double>());
   thrust::transform(d_particleDelta.begin(), d_particleDelta.end(), thrust::make_constant_iterator(1), d_particleDelta.begin(), thrust::minus<double>());
   thrust::transform(d_particleDelta.begin(), d_particleDelta.end(), thrust::make_constant_iterator(externalForce), d_particleDelta.begin(), thrust::multiplies<double>());
 }

 void DPM2D::addExternalParticleForce() {
   long p_nDim(nDim);
   auto r = thrust::counting_iterator<long>(0);
 	 double *pDelta = thrust::raw_pointer_cast(&d_particleDelta[0]);
 	 double *pForce = thrust::raw_pointer_cast(&d_particleForce[0]);

   auto addExternalForce = [=] __device__ (long particleId) {
     pForce[particleId * p_nDim] += pDelta[particleId];
   };

   thrust::for_each(r, r + numParticles, addExternalForce);
 }

 thrust::host_vector<double> DPM2D::getExternalParticleForce() {
   // return signed external forces
   thrust::host_vector<double> particleExternalForce;
   particleExternalForce = d_particleDelta;
   return particleExternalForce;
 }

 // return the sum of force magnitudes
 double DPM2D::getParticleTotalForceMagnitude() {
   thrust::device_vector<double> forceSquared(d_force.size());
   // compute squared velocities
   thrust::transform(d_particleForce.begin(), d_particleForce.end(), forceSquared.begin(), square());
   // sum squares
   double totalForceMagnitude = sqrt(thrust::reduce(forceSquared.begin(), forceSquared.end(), double(0), thrust::plus<double>()) / (numParticles * nDim));
   forceSquared.clear();
   return totalForceMagnitude;
 }

double DPM2D::getParticleMaxUnbalancedForce() {
  thrust::device_vector<double> forceSquared(d_particleForce.size());
  thrust::transform(d_particleForce.begin(), d_particleForce.end(), forceSquared.begin(), square());
  double maxUnbalancedForce = sqrt(thrust::reduce(forceSquared.begin(), forceSquared.end(), double(-1), thrust::maximum<double>()));
  forceSquared.clear();
  return maxUnbalancedForce;
}

double DPM2D::getRigidMaxUnbalancedForce() {
  //calcRigidForceEnergy();
  thrust::device_vector<double> forceSquared(d_particleForce.size());
  thrust::transform(d_particleForce.begin(), d_particleForce.end(), forceSquared.begin(), square());
  double particleMaxUnbalancedForce = sqrt(thrust::reduce(forceSquared.begin(), forceSquared.end(), double(-1), thrust::maximum<double>()));
	forceSquared.resize(d_particleTorque.size());
	thrust::transform(d_particleTorque.begin(), d_particleTorque.end(), forceSquared.begin(), square());
	double particleMaxUnbalancedTorque = sqrt(thrust::reduce(forceSquared.begin(), forceSquared.end(), double(-1), thrust::maximum<double>()));
  forceSquared.clear();
	return std::max(particleMaxUnbalancedForce, particleMaxUnbalancedTorque);
}

void DPM2D::calcParticleStressTensor() {
  thrust::fill(d_stress.begin(), d_stress.end(), double(0));
  const double *pRad = thrust::raw_pointer_cast(&d_particleRad[0]);
  const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
  double *pStress = thrust::raw_pointer_cast(&d_stress[0]);
  kernelCalcParticleStressTensor<<<partDimGrid, dimBlock>>>(pRad, pPos, pStress);
}

double DPM2D::getParticleVirialPressure() {
   calcParticleStressTensor();
	 double totalStress = 0, volume = 1;
	 for (long dim = 0; dim < nDim; dim++) {
		 totalStress += d_stress[dim * nDim + dim];
     volume *= d_boxSize[dim];
	 }
	 return totalStress / (nDim * volume);
	 //return totalStress;
}

double DPM2D::getParticleDynamicalPressure() {
  double volume = 1;
  for (long dim = 0; dim < nDim; dim++) {
    volume *= d_boxSize[dim];
  }
  return getParticleTemperature() * numParticles / volume;
}

double DPM2D::getParticleWallPressure() {
	 double wallWork = 0, volume = 1;
	 for (long dim = 0; dim < nDim; dim++) {
     volume *= d_boxSize[dim];
	 }
   const double *pRad = thrust::raw_pointer_cast(&d_particleRad[0]);
   const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
   kernelCalcParticleWallPressure<<<partDimGrid, dimBlock>>>(pRad, pPos, wallWork);
	 return wallWork / (nDim * volume);
	 //return totalStress;
}

double DPM2D::getParticleActivePressure(double driving) {
  double activeWork = 0, volume = 1;
  for (long dim = 0; dim < nDim; dim++) {
    volume *= d_boxSize[dim];
  }
	const double *pAngle = thrust::raw_pointer_cast(&d_particleAngle[0]);
	const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
  kernelCalcParticleActivePressure<<<partDimGrid, dimBlock>>>(pAngle, pPos, driving, activeWork);

  return activeWork / (nDim * volume);
}

double DPM2D::getParticleTotalPressure(double driving) {
  return getParticleDynamicalPressure() + getParticleActivePressure(driving);
}

double DPM2D::getParticleEnergy() {
  return thrust::reduce(d_particleEnergy.begin(), d_particleEnergy.end(), double(0), thrust::plus<double>());
}

double DPM2D::getParticleKineticEnergy() {
  thrust::device_vector<double> velSquared(d_particleVel.size());
  // compute squared velocities
  thrust::transform(d_particleVel.begin(), d_particleVel.end(), velSquared.begin(), square());
  // sum squares
  //cout << "vel squared: " << velSquared[0] << " " << velSquared[1] << " " << thrust::reduce(velSquared.begin(), velSquared.end(), double(0), thrust::plus<double>()) << endl;
  return 0.5 * thrust::reduce(velSquared.begin(), velSquared.end(), double(0), thrust::plus<double>());
}

double DPM2D::getParticleTemperature() {
  double ekin = getParticleKineticEnergy();
  return 2 * ekin / (numParticles * nDim);
}

double DPM2D::getMassiveTemperature(long firstIndex, double mass) {
  // temperature computed from the massive particles which are set to be the first #
  thrust::device_vector<double> velSquared(firstIndex * nDim);
  // compute squared velocities
  thrust::transform(d_particleVel.begin(), d_particleVel.begin() + firstIndex * nDim, velSquared.begin(), square());
  return mass * thrust::reduce(velSquared.begin(), velSquared.end(), double(0), thrust::plus<double>()) / (firstIndex * nDim);
}

double DPM2D::getParticleDrift() {
  return thrust::reduce(d_particlePos.begin(), d_particlePos.end(), double(0), thrust::plus<double>()) / (numParticles * nDim);
}

thrust::host_vector<long> DPM2D::getParticleNeighbors() {
  thrust::host_vector<long> partNeighborListFromDevice;
  partNeighborListFromDevice = d_partNeighborList;
  return partNeighborListFromDevice;
}

void DPM2D::calcParticleNeighborList(double cutDistance) {
  thrust::fill(d_partMaxNeighborList.begin(), d_partMaxNeighborList.end(), 0);
	thrust::fill(d_partNeighborList.begin(), d_partNeighborList.end(), -1L);
  syncParticleNeighborsToDevice();
  const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
	const double *pRad = thrust::raw_pointer_cast(&d_particleRad[0]);

  kernelCalcParticleNeighborList<<<partDimGrid, dimBlock>>>(pPos, pRad, cutDistance);
  // compute maximum number of neighbors per particle
  partMaxNeighbors = thrust::reduce(d_partMaxNeighborList.begin(), d_partMaxNeighborList.end(), -1L, thrust::maximum<long>());
  syncParticleNeighborsToDevice();
  //cout << "DPM2D::calcParticleNeighborList: maxNeighbors: " << partMaxNeighbors << endl;

  // if the neighbors don't fit, resize the neighbor list
  if ( partMaxNeighbors > partNeighborListSize ) {
		partNeighborListSize = pow(2, ceil(std::log2(partMaxNeighbors)));
    //cout << "DPM2D::calcParticleNeighborList: neighborListSize: " << neighborListSize << endl;
		//Now create the actual storage and then put the neighbors in it.
		d_partNeighborList.resize(numParticles * partNeighborListSize);
		//Pre-fill the neighborList with -1
		thrust::fill(d_partNeighborList.begin(), d_partNeighborList.end(), -1L);
		syncParticleNeighborsToDevice();
		kernelCalcParticleNeighborList<<<partDimGrid, dimBlock>>>(pPos, pRad, cutDistance);
	}
}

void DPM2D::syncParticleNeighborsToDevice() {
	//Copy the pointers and information about neighbors to the gpu
	hipMemcpyToSymbol(HIP_SYMBOL(d_partNeighborListSize), &partNeighborListSize, sizeof(partNeighborListSize));
	hipMemcpyToSymbol(HIP_SYMBOL(d_partMaxNeighbors), &partMaxNeighbors, sizeof(partMaxNeighbors));

	long* partMaxNeighborList = thrust::raw_pointer_cast(&d_partMaxNeighborList[0]);
	hipMemcpyToSymbol(HIP_SYMBOL(d_partMaxNeighborListPtr), &partMaxNeighborList, sizeof(partMaxNeighborList));

	long* partNeighborList = thrust::raw_pointer_cast(&d_partNeighborList[0]);
	hipMemcpyToSymbol(HIP_SYMBOL(d_partNeighborListPtr), &partNeighborList, sizeof(partNeighborList));
}

void DPM2D::calcParticleWallNeighborList(double cutDistance) {
  thrust::fill(d_partMaxNeighborList.begin(), d_partMaxNeighborList.end(), 0);
	thrust::fill(d_partNeighborList.begin(), d_partNeighborList.end(), -1L);
  syncParticleNeighborsToDevice();
  const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
	const double *pRad = thrust::raw_pointer_cast(&d_particleRad[0]);

  kernelCalcParticleWallNeighborList<<<partDimGrid, dimBlock>>>(pPos, pRad, cutDistance);
  // compute maximum number of neighbors per particle
  partMaxNeighbors = thrust::reduce(d_partMaxNeighborList.begin(), d_partMaxNeighborList.end(), -1L, thrust::maximum<long>());
  syncParticleNeighborsToDevice();
  //cout << "DPM2D::calcParticleNeighborList: maxNeighbors: " << partMaxNeighbors << endl;

  // if the neighbors don't fit, resize the neighbor list
  if ( partMaxNeighbors > partNeighborListSize ) {
		partNeighborListSize = pow(2, ceil(std::log2(partMaxNeighbors)));
    //cout << "DPM2D::calcParticleNeighborList: neighborListSize: " << neighborListSize << endl;
		//Now create the actual storage and then put the neighbors in it.
		d_partNeighborList.resize(numParticles * partNeighborListSize);
		//Pre-fill the neighborList with -1
		thrust::fill(d_partNeighborList.begin(), d_partNeighborList.end(), -1L);
		syncParticleNeighborsToDevice();
		kernelCalcParticleNeighborList<<<partDimGrid, dimBlock>>>(pPos, pRad, cutDistance);
	}
}

void DPM2D::calcParticleContacts(double gapSize) {
  long largestContact = 8*nDim; // Guess
	do {
		//Make a contactList that is the right size
		contactLimit = largestContact;
		d_contactList = thrust::device_vector<long>(numParticles * contactLimit);
		//Prefill the contactList with -1
		thrust::fill(d_contactList.begin(), d_contactList.end(), -1L);
		thrust::fill(d_numContacts.begin(), d_numContacts.end(), -1L);
		//Create device_pointers from thrust arrays
    const double *pPos = thrust::raw_pointer_cast(&d_particlePos[0]);
  	const double *pRad = thrust::raw_pointer_cast(&d_particleRad[0]);
		long* contactList = thrust::raw_pointer_cast(&d_contactList[0]);
		long* numContacts = thrust::raw_pointer_cast(&d_numContacts[0]);
		kernelCalcParticleContacts<<<dimGrid, dimBlock>>>(pPos, pRad, gapSize, contactLimit, contactList, numContacts);
		//Calculate the maximum number of contacts
		largestContact = thrust::reduce(d_numContacts.begin(), d_numContacts.end(), -1L, thrust::maximum<long>());
    //cout << "DPM2D::calcParticleContacts: largestContact = " << largestContact << endl;
	} while(contactLimit < largestContact); // If the guess was not good, do it again
}

//************************** minimizer functions *****************************//
void DPM2D::initFIRE(std::vector<double> &FIREparams, long minStep_, long numStep_, long numDOF_) {
  this->fire_ = new FIRE(this);
  if(FIREparams.size() == 7) {
    double a_start_ = FIREparams[0];
    double f_dec_ = FIREparams[1];
    double f_inc_ = FIREparams[2];
    double f_a_ = FIREparams[3];
    double fire_dt_ = FIREparams[4];
    double fire_dt_max_ = FIREparams[5];
    double a_ = FIREparams[6];
    this->fire_->initMinimizer(a_start_, f_dec_, f_inc_, f_a_, fire_dt_, fire_dt_max_, a_, minStep_, numStep_, numDOF_);
  } else {
    cout << "DPM2D::initFIRE: wrong number of FIRE parameters, must be 7" << endl;
  }
}

void DPM2D::setParticleMassFIRE() {
  //this->fire_->setParticleMass();
  this->fire_->d_mass.resize(numParticles * nDim);
	for (long particleId = 0; particleId < numParticles; particleId++) {
		for (long dim = 0; dim < nDim; dim++) {
			this->fire_->d_mass[particleId * nDim + dim] = PI / (d_particleRad[particleId] * d_particleRad[particleId]);
		}
	}
}

void DPM2D::setTimeStepFIRE(double timeStep_) {
  this->fire_->setFIRETimeStep(timeStep_);
}


void DPM2D::particleFIRELoop() {
  this->fire_->minimizerParticleLoop();
}

void DPM2D::vertexFIRELoop() {
  this->fire_->minimizerVertexLoop();
}

void DPM2D::initRigidFIRE(std::vector<double> &FIREparams, long minStep_, long numStep_, long numDOF_, double cutDist_) {
  initFIRE(FIREparams, minStep_, numStep_, numDOF_);
  initDeltaVariables(getNumVertices(), getNumParticles());
  initRotationalVariables(getNumVertices(), getNumParticles());
  this->fire_->cutDistance = cutDist_;
}

void DPM2D::rigidFIRELoop() {
  this->fire_->minimizerRigidLoop();
}

//********************* deformable particles integrators *********************//
void DPM2D::initLangevin(double Temp, double gamma, bool readState) {
  this->sim_ = new Langevin2(this, SimConfig(Temp, 0, 0, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->lcoeff1 = 0.25 * dt * sqrt(dt) * gamma * this->sim_->noiseVar;
  this->sim_->lcoeff2 = 0.5 * sqrt(dt) * this->sim_->noiseVar;
  this->sim_->lcoeff3 = (0.5 / sqrt(3)) * sqrt(dt) * dt * this->sim_->noiseVar;
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_rando.resize(numParticles * nDim);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  thrust::fill(this->sim_->d_thermalVel.begin(), this->sim_->d_thermalVel.end(), double(0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  calcParticlesPositions();
  d_particleInitPos = getParticlePositions();
  //cout << "DPM2D::initLangevin:: damping coefficients: " << this->sim_->lcoeff1 << " " << this->sim_->lcoeff2 << " " << this->sim_->lcoeff3 << endl;
  if(readState == false) {
    //this->sim_->injectKineticEnergy();
    thrust::fill(d_vel.begin(), d_vel.end(), double(0));
  }
  cout << "DPM2D::initLangevin:: current temperature: " << setprecision(12) << getTemperature() << endl;
}

void DPM2D::langevinLoop() {
  this->sim_->integrate();
}

void DPM2D::initActiveLangevin(double Temp, double Dr, double driving, double gamma, bool readState) {
  this->sim_ = new ActiveLangevin(this, SimConfig(Temp, Dr, driving, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->lcoeff1 = 0.25 * dt * sqrt(dt) * gamma * this->sim_->noiseVar;
  this->sim_->lcoeff2 = 0.5 * sqrt(dt) * this->sim_->noiseVar;
  this->sim_->lcoeff3 = (0.5 / sqrt(3)) * sqrt(dt) * dt * this->sim_->noiseVar;
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_rando.resize(numParticles * nDim);
  this->sim_->d_pActiveAngle.resize(numParticles);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  thrust::fill(this->sim_->d_thermalVel.begin(), this->sim_->d_thermalVel.end(), double(0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  calcParticlesPositions();
  d_particleInitPos = getParticlePositions();
  //cout << "DPM2D::initActiveLangevin:: damping coefficients: " << this->sim_->lcoeff1 << " " << this->sim_->lcoeff2 << " " << this->sim_->lcoeff3 << endl;
  if(readState == false) {
    //this->sim_->injectKineticEnergy();
    thrust::fill(d_vel.begin(), d_vel.end(), double(0));
  }
  cout << "DPM2D::initActiveLangevin:: current temperature: " << setprecision(12) << getTemperature() << endl;
}

void DPM2D::activeLangevinLoop() {
  this->sim_->integrate();
}

void DPM2D::initNVE(double Temp, bool readState) {
  this->sim_ = new NVE(this, SimConfig(Temp, 0, 0, 0));
  this->sim_->noiseVar = sqrt(2. * Temp);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  if(readState == false) {
    this->sim_->injectKineticEnergy();
  }
  cout << "DPM2D::initNVE:: current temperature: " << setprecision(12) << getTemperature() << endl;
}

void DPM2D::NVELoop() {
  this->sim_->integrate();
}

void DPM2D::initBrownian(double Temp, double gamma, bool readState) {
  this->sim_ = new Brownian(this, SimConfig(Temp, 0, 0, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  if(readState == false) {
    this->sim_->injectKineticEnergy();
  }
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  cout << "DPM2D::initBrownian:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::brownianLoop() {
  this->sim_->integrate();
}

void DPM2D::initActiveBrownian(double Dr, double driving, bool readState) {
  this->sim_ = new ActiveBrownian(this, SimConfig(0, Dr, driving, 0));
  this->sim_->d_rand.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    computeParticleAngleFromVel();
    //thrust::counting_iterator<long> index_sequence_begin(drand48());
    //thrust::transform(index_sequence_begin, index_sequence_begin + numParticles, d_particleAngle.begin(), randNum(0.f, 2.f * PI));
  }
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  cout << "DPM2D::initActiveBrownian:: current temperature: " << setprecision(12) << getTemperature() << endl;
}

void DPM2D::activeBrownianLoop() {
  this->sim_->integrate();
}

void DPM2D::initActiveBrownianDampedL0(double Dr, double driving, double gamma, bool readState) {
  this->sim_ = new ActiveBrownianDampedL0(this, SimConfig(0, Dr, driving, 0));
  this->sim_->lcoeff1 = exp(-gamma * dt);
  this->sim_->d_rand.resize(numParticles);
  d_l0Vel.resize(numVertices);
  thrust::fill(d_l0Vel.begin(), d_l0Vel.end(), double(0));
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    computeParticleAngleFromVel();
    //thrust::counting_iterator<long> index_sequence_begin(drand48());
    //thrust::transform(index_sequence_begin, index_sequence_begin + numParticles, d_particleAngle.begin(), randNum(0.f, 2.f * PI));
  }
  cout << "DPM2D::initActiveBrownian:: current temperature: " << setprecision(12) << getTemperature() << endl;
}

void DPM2D::activeBrownianDampedL0Loop() {
  this->sim_->integrate();
}

//************************* soft particle simulators *************************//
void DPM2D::computeParticleDrift() {
  thrust::fill(d_delta.begin(), d_delta.end(), double(0));
  double *velSum = thrust::raw_pointer_cast(&d_delta[0]);
  const double *pVel = thrust::raw_pointer_cast(&d_particleVel[0]);
  kernelSumParticleVelocity<<<partDimGrid, dimBlock>>>(pVel, velSum);
}

void DPM2D::conserveParticleMomentum() {
  double *pVel = thrust::raw_pointer_cast(&d_particleVel[0]);
  const double *velSum = thrust::raw_pointer_cast(&d_delta[0]);
  kernelSubtractParticleDrift<<<partDimGrid, dimBlock>>>(pVel, velSum);
}

void DPM2D::initSoftParticleLangevin(double Temp, double gamma, bool readState) {
  this->sim_ = new SoftParticleLangevin2(this, SimConfig(Temp, 0, 0, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->lcoeff1 = 0.25 * dt * sqrt(dt) * gamma * this->sim_->noiseVar;
  this->sim_->lcoeff2 = 0.5 * sqrt(dt) * this->sim_->noiseVar;
  this->sim_->lcoeff3 = (0.5 / sqrt(3)) * sqrt(dt) * dt * this->sim_->noiseVar;
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_rando.resize(numParticles * nDim);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  thrust::fill(this->sim_->d_thermalVel.begin(), this->sim_->d_thermalVel.end(), double(0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  //d_delta.resize(nDim);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    //cout << "DPM2D::initSoftParticleLangevin:: damping coefficients: " << this->sim_->lcoeff1 << " " << this->sim_->lcoeff2 << " " << this->sim_->lcoeff3 << endl;
  }
  cout << "DPM2D::initSoftParticleLangevin:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::softParticleLangevinLoop() {
  this->sim_->integrate();
  //computeParticleDrift();
  //conserveParticleMomentum();
  //computeParticleDrift();
  //cout << "velSum: " << thrust::reduce(d_particleVel.begin(), d_particleVel.end(), double(0), thrust::plus<double>()) << endl;
}

void DPM2D::initSoftParticleLangevinFixedBoundary(double Temp, double gamma, bool readState) {
  this->sim_ = new SoftParticleLangevinFixedBoundary(this, SimConfig(Temp, 0, 0, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->lcoeff1 = 0.25 * dt * sqrt(dt) * gamma * this->sim_->noiseVar;
  this->sim_->lcoeff2 = 0.5 * sqrt(dt) * this->sim_->noiseVar;
  this->sim_->lcoeff3 = (0.5 / sqrt(3)) * sqrt(dt) * dt * this->sim_->noiseVar;
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_rando.resize(numParticles * nDim);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  thrust::fill(this->sim_->d_thermalVel.begin(), this->sim_->d_thermalVel.end(), double(0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    //cout << "DPM2D::initSoftParticleLangevin:: damping coefficients: " << this->sim_->lcoeff1 << " " << this->sim_->lcoeff2 << " " << this->sim_->lcoeff3 << endl;
  }
  cout << "DPM2D::initSoftParticleLangevinFixedBoundary:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::softParticleLangevinFixedBoundaryLoop() {
  this->sim_->integrate();
}

void DPM2D::initSoftParticleNVE(double Temp, bool readState) {
  this->sim_ = new SoftParticleNVE(this, SimConfig(Temp, 0, 0, 0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
  }
  cout << "DPM2D::initSoftParticleNVE:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::softParticleNVELoop() {
  this->sim_->integrate();
}

void DPM2D::initSoftParticleNVERA(double Temp, bool readState) {
  this->sim_ = new SoftParticleNVERA(this, SimConfig(Temp, 0, 0, 0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
  }
  cout << "DPM2D::initSoftParticleNVERA:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::softParticleNVERALoop() {
  this->sim_->integrate();
}

void DPM2D::initSoftParticleNVEFixedBoundary(double Temp, bool readState) {
  this->sim_ = new SoftParticleNVEFixedBoundary(this, SimConfig(Temp, 0, 0, 0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
  }
  cout << "DPM2D::initSoftParticleNVEFixedBoundary:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::softParticleNVEFixedBoundaryLoop() {
  this->sim_->integrate();
}

void DPM2D::initSoftParticleActiveNVEFixedBoundary(double Temp, double Dr, double driving, bool readState) {
  this->sim_ = new SoftParticleActiveNVEFixedBoundary(this, SimConfig(Temp, 0, 0, 0));
  this->sim_->d_pActiveAngle.resize(numParticles);
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    computeParticleAngleFromVel();
  }
  cout << "DPM2D::initSoftParticleActiveNVEFixedBoundary:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::softParticleActiveNVEFixedBoundaryLoop() {
  d_particlePreviousPos = getParticlePositions();
  this->sim_->integrate();
}

void DPM2D::initSoftParticleLangevinRA(double Temp, double gamma, bool readState) {
  this->sim_ = new SoftParticleLangevin2RA(this, SimConfig(Temp, 0, 0, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->lcoeff1 = 0.25 * dt * sqrt(dt) * gamma * this->sim_->noiseVar;
  this->sim_->lcoeff2 = 0.5 * sqrt(dt) * this->sim_->noiseVar;
  this->sim_->lcoeff3 = (0.5 / sqrt(3)) * sqrt(dt) * dt * this->sim_->noiseVar;
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_rando.resize(numParticles * nDim);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  thrust::fill(this->sim_->d_thermalVel.begin(), this->sim_->d_thermalVel.end(), double(0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    //cout << "DPM2D::initSoftParticleLangevin:: damping coefficients: " << this->sim_->lcoeff1 << " " << this->sim_->lcoeff2 << " " << this->sim_->lcoeff3 << endl;
  }
  cout << "DPM2D::initSoftParticleLangevinRA:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::softParticleLangevinRALoop() {
  this->sim_->integrate();
}

void DPM2D::initSoftParticleLangevinLJ(double Temp, double gamma, bool readState) {
  this->sim_ = new SoftParticleLangevin2LJ(this, SimConfig(Temp, 0, 0, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->lcoeff1 = 0.25 * dt * sqrt(dt) * gamma * this->sim_->noiseVar;
  this->sim_->lcoeff2 = 0.5 * sqrt(dt) * this->sim_->noiseVar;
  this->sim_->lcoeff3 = (0.5 / sqrt(3)) * sqrt(dt) * dt * this->sim_->noiseVar;
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_rando.resize(numParticles * nDim);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  thrust::fill(this->sim_->d_thermalVel.begin(), this->sim_->d_thermalVel.end(), double(0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    //cout << "DPM2D::initSoftParticleLangevin:: damping coefficients: " << this->sim_->lcoeff1 << " " << this->sim_->lcoeff2 << " " << this->sim_->lcoeff3 << endl;
  }
  cout << "DPM2D::initSoftParticleLangevinLJ:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::softParticleLangevinLJLoop() {
  this->sim_->integrate();
}

void DPM2D::initSoftLangevinSubSet(double Temp, double gamma, long firstIndex, double mass, bool readState, bool zeroOutMassiveVel) {
  this->sim_ = new SoftLangevinSubSet(this, SimConfig(Temp, 0, 0, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->lcoeff1 = 0.25 * dt * sqrt(dt) * gamma * this->sim_->noiseVar;
  this->sim_->lcoeff2 = 0.5 * sqrt(dt) * this->sim_->noiseVar;
  this->sim_->lcoeff3 = (0.5 / sqrt(3)) * sqrt(dt) * dt * this->sim_->noiseVar;
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_rando.resize(numParticles * nDim);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  // subset variables
  this->sim_->firstIndex = firstIndex;
  this->sim_->mass = mass;
  thrust::fill(this->sim_->d_thermalVel.begin(), this->sim_->d_thermalVel.end(), double(0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
  }
  if(zeroOutMassiveVel == true) {
    thrust::fill(d_particleVel.begin(), d_particleVel.begin() + firstIndex * nDim, double(0));
  }
  cout << "DPM2D::initSoftLangevinSubSet:: current temperature: " << setprecision(12) << getParticleTemperature() << " mass: " << this->sim_->mass << endl;
}

void DPM2D::softLangevinSubSetLoop() {
  this->sim_->integrate();
}

void DPM2D::initSoftParticleLExtField(double Temp, double gamma, bool readState) {
  this->sim_ = new SoftParticleLExtField(this, SimConfig(Temp, 0, 0, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->lcoeff1 = 0.25 * dt * sqrt(dt) * gamma * this->sim_->noiseVar;
  this->sim_->lcoeff2 = 0.5 * sqrt(dt) * this->sim_->noiseVar;
  this->sim_->lcoeff3 = (0.5 / sqrt(3)) * sqrt(dt) * dt * this->sim_->noiseVar;
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_rando.resize(numParticles * nDim);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  thrust::fill(this->sim_->d_thermalVel.begin(), this->sim_->d_thermalVel.end(), double(0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
  }
  cout << "DPM2D::initSoftParticleLangevin:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::softParticleLExtFieldLoop() {
  this->sim_->integrate();
}

void DPM2D::initSoftParticleActiveLangevin(double Temp, double Dr, double driving, double gamma, bool readState) {
  this->sim_ = new SoftParticleActiveLangevin(this, SimConfig(Temp, Dr, driving, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->lcoeff1 = 0.25 * dt * sqrt(dt) * gamma * this->sim_->noiseVar;
  this->sim_->lcoeff2 = 0.5 * sqrt(dt) * this->sim_->noiseVar;
  this->sim_->lcoeff3 = (0.5 / sqrt(3)) * sqrt(dt) * dt * this->sim_->noiseVar;
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_rando.resize(numParticles * nDim);
  this->sim_->d_pActiveAngle.resize(numParticles);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  thrust::fill(this->sim_->d_thermalVel.begin(), this->sim_->d_thermalVel.end(), double(0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    computeParticleAngleFromVel();
    //cout << "DPM2D::initSoftParticleActiveLangevin:: damping coefficients: " << this->sim_->lcoeff1 << " " << this->sim_->lcoeff2 << " " << this->sim_->lcoeff3 << endl;
  }
  cout << "DPM2D::initSoftParticleActiveLangevin:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::softParticleActiveLangevinLoop() {
  //d_particlePreviousPos = getParticlePositions();
  this->sim_->integrate();
}

void DPM2D::initSoftParticleActiveLangevinFixedBoundary(double Temp, double Dr, double driving, double gamma, bool readState) {
  this->sim_ = new SoftParticleActiveLangevinFixedBoundary(this, SimConfig(Temp, Dr, driving, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->lcoeff1 = 0.25 * dt * sqrt(dt) * gamma * this->sim_->noiseVar;
  this->sim_->lcoeff2 = 0.5 * sqrt(dt) * this->sim_->noiseVar;
  this->sim_->lcoeff3 = (0.5 / sqrt(3)) * sqrt(dt) * dt * this->sim_->noiseVar;
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_rando.resize(numParticles * nDim);
  this->sim_->d_pActiveAngle.resize(numParticles);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  thrust::fill(this->sim_->d_thermalVel.begin(), this->sim_->d_thermalVel.end(), double(0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    computeParticleAngleFromVel();
  }
  cout << "DPM2D::initSoftParticleActiveLangevinFixedBoundary:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::softParticleActiveLangevinFixedBoundaryLoop() {
  this->sim_->integrate();

}

void DPM2D::initSoftParticleActiveLangevinFixedSides(double Temp, double Dr, double driving, double gamma, bool readState) {
  this->sim_ = new SoftParticleActiveLangevinFixedSides(this, SimConfig(Temp, Dr, driving, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->lcoeff1 = 0.25 * dt * sqrt(dt) * gamma * this->sim_->noiseVar;
  this->sim_->lcoeff2 = 0.5 * sqrt(dt) * this->sim_->noiseVar;
  this->sim_->lcoeff3 = (0.5 / sqrt(3)) * sqrt(dt) * dt * this->sim_->noiseVar;
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_rando.resize(numParticles * nDim);
  this->sim_->d_pActiveAngle.resize(numParticles);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  thrust::fill(this->sim_->d_thermalVel.begin(), this->sim_->d_thermalVel.end(), double(0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    computeParticleAngleFromVel();
  }
  cout << "DPM2D::initSoftParticleActiveLangevinFixedSides:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::softParticleActiveLangevinFixedSidesLoop() {
  this->sim_->integrate();

}

void DPM2D::initSoftALSubSet(double Temp, double Dr, double driving, double gamma, long firstIndex, double mass, bool readState, bool zeroOutMassiveVel) {
  this->sim_ = new SoftALSubSet(this, SimConfig(Temp, Dr, driving, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->lcoeff1 = 0.25 * dt * sqrt(dt) * gamma * this->sim_->noiseVar;
  this->sim_->lcoeff2 = 0.5 * sqrt(dt) * this->sim_->noiseVar;
  this->sim_->lcoeff3 = (0.5 / sqrt(3)) * sqrt(dt) * dt * this->sim_->noiseVar;
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_rando.resize(numParticles * nDim);
  this->sim_->d_pActiveAngle.resize(numParticles);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  // subset variables
  this->sim_->firstIndex = firstIndex;
  this->sim_->mass = mass;
  thrust::fill(this->sim_->d_thermalVel.begin(), this->sim_->d_thermalVel.end(), double(0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    computeParticleAngleFromVel();
  }
  if(zeroOutMassiveVel == true) {
    thrust::fill(d_particleVel.begin(), d_particleVel.begin() + firstIndex * nDim, double(0));
  }
  cout << "DPM2D::initSoftALSubSet:: current temperature: " << setprecision(12) << getParticleTemperature() << " mass: " << this->sim_->mass << endl;
}

void DPM2D::softALSubSetLoop() {
  this->sim_->integrate();
}

void DPM2D::initSoftParticleALExtField(double Temp, double Dr, double driving, double gamma, bool readState) {
  this->sim_ = new SoftParticleALExtField(this, SimConfig(Temp, Dr, driving, 0));
  this->sim_->gamma = gamma;
  this->sim_->noiseVar = sqrt(2. * Temp * gamma);
  this->sim_->lcoeff1 = 0.25 * dt * sqrt(dt) * gamma * this->sim_->noiseVar;
  this->sim_->lcoeff2 = 0.5 * sqrt(dt) * this->sim_->noiseVar;
  this->sim_->lcoeff3 = (0.5 / sqrt(3)) * sqrt(dt) * dt * this->sim_->noiseVar;
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_rando.resize(numParticles * nDim);
  this->sim_->d_pActiveAngle.resize(numParticles);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  thrust::fill(this->sim_->d_thermalVel.begin(), this->sim_->d_thermalVel.end(), double(0));
  d_particleInitPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particleInitPos = getParticlePositions();
  d_particlePreviousPos.resize(numParticles * nDim);
  thrust::fill(d_particleInitPos.begin(), d_particleInitPos.end(), double(0));
  d_particlePreviousPos = getParticlePositions();
  d_particleDelta.resize(numParticles * nDim);
  d_particleDisp.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    computeParticleAngleFromVel();
  }
  cout << "DPM2D::initSoftParticleALExtField:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::softParticleALExtFieldLoop() {
  this->sim_->integrate();
}


//************************* rigid particle simulators ************************//
void DPM2D::initRigidBrownian(double Temp, double cutDistance, bool readState) {
  this->sim_ = new RigidBrownian(this, SimConfig(Temp, 0, 0, cutDistance));
  this->sim_->d_rand.resize(numParticles * nDim);
  this->sim_->d_thermalVel.resize(d_particleVel.size());
  if(readState == false) {
      this->sim_->injectKineticEnergy();
  }
  initDeltaVariables(getNumVertices(), getNumParticles());
  initRotationalVariables(getNumVertices(), getNumParticles());
  d_initialPos = getVertexPositions();
  calcParticlesPositions();
  d_particleInitPos = getParticlePositions();
  cout << "DPM2D::initRigidBrownian:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::rigidBrownianLoop() {
  this->sim_->integrate();
}

void DPM2D::initRigidRotActiveBrownian(double Dr, double driving, double cutDistance, bool readState) {
  this->sim_ = new RigidRotActiveBrownian(this, SimConfig(0, Dr, driving, cutDistance));
  this->sim_->d_rand.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    computeParticleAngleFromVel();
    //thrust::counting_iterator<long> index_sequence_begin(drand48());
    //thrust::transform(index_sequence_begin, index_sequence_begin + numParticles, d_particleAngle.begin(), randNum(0.f, 2.f * PI));
  }
  initDeltaVariables(getNumVertices(), getNumParticles());
  initRotationalVariables(getNumVertices(), getNumParticles());
  d_initialPos = getVertexPositions();
  calcParticlesPositions();
  d_particleInitPos = getParticlePositions();
  cout << "DPM2D::initRigidRotActiveBrownian:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::rigidRotActiveBrownianLoop() {
  this->sim_->integrate();
}

void DPM2D::initRigidActiveBrownian(double Dr, double driving, double cutDistance, bool readState) {
  this->sim_ = new RigidActiveBrownian(this, SimConfig(0, Dr, driving, cutDistance));
  this->sim_->d_rand.resize(numParticles);
  this->sim_->d_pActiveAngle.resize(numParticles);
  if(readState == false) {
    this->sim_->injectKineticEnergy();
    computeParticleAngleFromVel();
    //thrust::counting_iterator<long> index_sequence_begin(drand48());
    //thrust::transform(index_sequence_begin, index_sequence_begin + numParticles, d_particleAngle.begin(), randNum(0.f, 2.f * PI));
  }
  initDeltaVariables(getNumVertices(), getNumParticles());
  initRotationalVariables(getNumVertices(), getNumParticles());
  d_initialPos = getVertexPositions();
  calcParticlesPositions();
  d_particleInitPos = getParticlePositions();
  cout << "DPM2D::initRigidActiveBrownian:: current temperature: " << setprecision(12) << getParticleTemperature() << endl;
}

void DPM2D::rigidActiveBrownianLoop() {
  this->sim_->integrate();
}
